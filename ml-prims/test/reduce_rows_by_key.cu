#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "kmeans/reduce_rows_by_key.h"
#include "random/rng.h"
#include "test_utils.h"
#include <iostream>

namespace MLCommon {
namespace KMeans {


template <typename Type>
__global__ void naiveReduceRowsByKeyKernel(Type *d_A, int lda,
                          int *d_keys,char *d_char_keys, int nrows, 
                          int ncols, int nkeys, Type *d_sums) 
{
    int c=threadIdx.x + blockIdx.x*blockDim.x;
    int this_key = threadIdx.y+blockIdx.y*blockDim.y;
    
    for (int r=0;r<nrows;r++) {
       if (this_key != d_keys[r]) continue;
       myAtomicAdd(&d_sums[this_key*ncols+c], d_A[lda*r+c]);
    }
}
template <typename Type>
void naiveReduceRowsByKey( int stream, Type* d_A, int lda, 
                          int *d_keys,char *d_char_keys, int nrows, 
                          int ncols, int nkeys, Type *d_sums) 
{
    hipMemset(d_sums, 0, sizeof(Type) * nkeys*ncols);

    naiveReduceRowsByKeyKernel
          <<<dim3(ncols/32,1),dim3(32,nkeys)>>>
                      (d_A,lda,d_keys,d_char_keys,nrows,ncols,nkeys,d_sums);
    //for (int r=0;r<nrows;r++) {
    //   int this_key = d_keys[r];
    //   std::cout << "this_key[" << r << "] = " << this_key << std::endl;std::cout.flush();
    //   for (int c=0;c<ncols;c++) {
    //      d_sums[this_key*ncols+c] += d_A[lda*r+c];
    //   }
    //}
}
template <typename DataType, typename KeyType>
void reduce_rows_by_key( int stream, DataType* d_A, int lda, 
                          KeyType *d_keys, char *d_char_keys, int nrows, 
                          int ncols, int nkeys, DataType *d_sums) 
{
    hipMemset(d_sums, 0, sizeof(DataType) * nkeys*ncols);

    naiveReduceRowsByKeyKernel
          <<<dim3(ncols/32,1),dim3(32,nkeys)>>>
                      (d_A,lda,d_keys,d_char_keys,nrows,ncols,nkeys,d_sums);
}


template <typename T>
struct ReduceRowsInputs {
    T tolerance;
    int rows;
    int cols;
    int nkeys;
    unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const ReduceRowsInputs<T>& dims) {
    return os;
}

template <typename T>
class ReduceRowTest: public ::testing::TestWithParam<ReduceRowsInputs<T> > {
protected:
    void SetUp() override {
        params = ::testing::TestWithParam<ReduceRowsInputs<T>>::GetParam();
        Random::Rng<T> r(params.seed);
        Random::Rng<int> r_int(params.seed);
        int rows = params.rows;
        int cols = params.cols;
        int nkeys = params.nkeys;
        allocate(in1, rows*cols);
        allocate(in2, rows);
        allocate(chars2, rows);
        allocate(out_ref, nkeys*cols);
        allocate(out, nkeys*cols);
        r.uniform(in1, rows*cols, T(-1.0), T(1.0));
        r_int.randInt(in2, rows, 0, nkeys);
        naiveReduceRowsByKey(0, in1, cols, in2, chars2,
                               rows, cols, nkeys, out_ref );
        reduce_rows_by_key(0, in1, cols, in2, chars2, 
                               rows, cols, nkeys, out );
        /*
        CUDA_CHECK(hipGetDeviceCount(&device_count));
        if (device_count > 1) {
        	T *h_in1 = (T *) malloc(len * sizeof(T));
        	T *h_in2 = (T *) malloc(len * sizeof(T));
        	updateHost(h_in1, in1, len);
        	updateHost(h_in2, in2, len);
        	addMGColSplitTest(h_in1, h_in2);
        	free(h_in1);
        	free(h_in2);
        }
        */
    }
/*
    void addMGColSplitTest(T *h_in1, T *h_in2) {
    	int n_gpus = 2;

    	TypeMG<T> d_in1[n_gpus];
    	TypeMG<T> d_in2[n_gpus];
    	TypeMG<T> d_out[n_gpus];

    	for (int i = 0; i < n_gpus; i++) {
    		d_in1[i].gpu_id = i;
    		d_in2[i].gpu_id = i;
    		d_out[i].gpu_id = i;
    		CUDA_CHECK(hipSetDevice(d_in1[i].gpu_id));
    		CUDA_CHECK(hipStreamCreate(&(d_in1[i].stream)));
    		d_in2[i].stream = d_in1[i].stream;
    		d_out[i].stream = d_in1[i].stream;
    	}

    	int len = params.len;
    	allocateMG(d_in1, n_gpus, 1, len, true, true, false);
    	allocateMG(d_in2, n_gpus, 1, len, true, true, false);
    	allocateMG(d_out, n_gpus, 1, len, true, true, false);

    	updateDeviceMG(d_in1, h_in1, n_gpus, false);
    	updateDeviceMG(d_in1, h_in1, n_gpus, false);

    	addMG(d_out, d_in1, d_in2, len, n_gpus, false);

    	T *h_out = (T *) malloc(len * sizeof(T));
    	updateHostMG(h_out, d_out, n_gpus, false);

    	streamSyncMG(d_in1, n_gpus);
    	streamDestroyGPUs(d_in1, n_gpus);

    	freeMG(d_in1, n_gpus);
    	freeMG(d_in2, n_gpus);
    	freeMG(d_out, n_gpus);

    	allocate(out_2, len);
    	updateDevice(out_2, h_out, len);

    	free(h_out);
    }
*/

    void TearDown() override {
        CUDA_CHECK(hipFree(in1));
        CUDA_CHECK(hipFree(in2));
        CUDA_CHECK(hipFree(chars2));
        CUDA_CHECK(hipFree(out_ref));
        CUDA_CHECK(hipFree(out));

        //if (device_count > 1) {
        // CUDA_CHECK(hipFree(out_2));
        //}
    }

protected:
    ReduceRowsInputs<T> params;
    T *in1, *out_ref, *out, *out_2;
    int *in2;
    char *chars2;
    int device_count = 0;
};

const std::vector<ReduceRowsInputs<float> > inputsf2 = {
    {0.000001f, 128, 32, 6, 1234ULL}
};

const std::vector<ReduceRowsInputs<double> > inputsd2 = {
    {0.00000001, 128, 32, 6, 1234ULL}
};

typedef ReduceRowTest<double> ReduceRowTestD;
TEST_P(ReduceRowTestD, Result){
    ASSERT_TRUE(devArrMatch(out_ref, out, params.cols,
                            CompareApprox<double>(params.tolerance)));

    //ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
     //                       CompareApprox<double>(params.tolerance)));

    //if (device_count > 1) {
   // 	ASSERT_TRUE(devArrMatch(out_ref, out_2, params.len,
    //	                    CompareApprox<double>(params.tolerance)));
    //}
}

typedef ReduceRowTest<float> ReduceRowTestF;
TEST_P(ReduceRowTestF, Result) {
    ASSERT_TRUE(devArrMatch(out_ref, out, params.cols,
                            CompareApprox<float>(params.tolerance)));

    //ASSERT_TRUE(devArrMatch(out_ref, in1, params.cols,
      //                      CompareApprox<float>(params.tolerance)));

    //if (device_count > 1) {
    //	ASSERT_TRUE(devArrMatch(out_ref, out_2, params.cols,
    //	                    CompareApprox<float>(params.tolerance)));
    //}
}

INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestD,
                        ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestF,
                        ::testing::ValuesIn(inputsf2));

} // end namespace LinAlg
} // end namespace MLCommon
