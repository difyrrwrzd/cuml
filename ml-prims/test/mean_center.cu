/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "stats/mean.h"
#include "stats/mean_center.h"
#include "random/rng.h"
#include "test_utils.h"
#include "matrix/math.h"


namespace MLCommon {
namespace Stats {

template <typename T>
struct MeanCenterInputs {
    T tolerance, mean;
    int rows, cols;
    bool sample, rowMajor;
    unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const MeanCenterInputs<T>& dims) {
    return os;
}

template <typename T>
class MeanCenterTest: public ::testing::TestWithParam<MeanCenterInputs<T> > {
protected:
    void SetUp() override {
        params = ::testing::TestWithParam<MeanCenterInputs<T>>::GetParam();
        Random::Rng<T> r(params.seed);
        int rows = params.rows, cols = params.cols;
        int len = rows * cols;
        allocate(data, len);
        allocate(mean_act, cols);
        r.normal(data, len, params.mean, (T)1.0);
        mean(mean_act, data, cols, rows, params.sample, params.rowMajor);
        meanCenter(data, mean_act, cols, rows, params.rowMajor);

        mean(mean_act, data, cols, rows, params.sample, params.rowMajor);
        Matrix::matrixVectorBinarySub(data, mean_act, rows, cols, false);
        mean(mean_act, data, cols, rows, params.sample, params.rowMajor);

        CUDA_CHECK(hipGetDeviceCount(&device_count));

        if (device_count > 1) {
            T *h_data = (T *) malloc(len * sizeof(T));
            updateHost(h_data, data, len);
            meanCenterMGColSplitTest(h_data);
            free(h_data);
        }
    }

    void meanCenterMGColSplitTest(T *h_data) {
        int n_gpus = 2;

        TypeMG<T> d_data[n_gpus];
        TypeMG<T> d_mu[n_gpus];

        for (int i = 0; i < n_gpus; i++) {
        	d_data[i].gpu_id = i;
        	d_mu[i].gpu_id = i;

        	CUDA_CHECK(hipSetDevice(d_data[i].gpu_id));
        	CUDA_CHECK(hipStreamCreate(&(d_data[i].stream)));
        	d_mu[i].stream = d_data[i].stream;
        }

        allocateMG(d_data, n_gpus, params.rows, params.cols, true, true, false);
       	allocateMG(d_mu, n_gpus, 1, params.cols, true, true, false);

        updateDeviceMG(d_data, h_data, n_gpus, false);

        meanMG(d_mu, d_data, params.cols, params.rows, n_gpus, true, false,
        		false, false);

        meanCenterMG(d_data, d_mu, params.cols, params.rows, n_gpus, false,
        		false, false);

        meanMG(d_mu, d_data, params.cols, params.rows, n_gpus, true, false,
                		false, false);

        int len = params.cols;
        T *h_mu = (T *) malloc(len * sizeof(T));
        updateHostMG(h_mu, d_mu, n_gpus, false);

        streamSyncMG(d_data, n_gpus);
        streamDestroyGPUs(d_data, n_gpus);

        freeMG(d_data, n_gpus);
        freeMG(d_mu, n_gpus);

        allocate(mean_act_2, len);
        updateDevice(mean_act_2, h_mu, len);

        free(h_mu);
    }

    void TearDown() override {
        CUDA_CHECK(hipFree(data));
        CUDA_CHECK(hipFree(mean_act));
        if (device_count > 1) {
        	CUDA_CHECK(hipFree(mean_act_2));
        }
    }

protected:
    MeanCenterInputs<T> params;
    T *data, *mean_act, *mean_act_2;
    int device_count = 0;
};

const std::vector<MeanCenterInputs<float> > inputsf = {
    {0.05f,  1.f, 1024,  32,  true, false, 1234ULL},
    {0.05f,  1.f, 1024,  64,  true, false, 1234ULL},
    {0.05f,  1.f, 1024, 128,  true, false, 1234ULL},
    {0.05f,  1.f, 1024, 256,  true, false, 1234ULL},
    {0.05f, -1.f, 1024,  32, false, false, 1234ULL},
    {0.05f, -1.f, 1024,  64, false, false, 1234ULL},
    {0.05f, -1.f, 1024, 128, false, false, 1234ULL},
    {0.05f, -1.f, 1024, 256, false, false, 1234ULL},
    {0.05f,  1.f, 1024,  32,  true,  true, 1234ULL},
    {0.05f,  1.f, 1024,  64,  true,  true, 1234ULL},
    {0.05f,  1.f, 1024, 128,  true,  true, 1234ULL},
    {0.05f,  1.f, 1024, 256,  true,  true, 1234ULL},
    {0.05f, -1.f, 1024,  32, false,  true, 1234ULL},
    {0.05f, -1.f, 1024,  64, false,  true, 1234ULL},
    {0.05f, -1.f, 1024, 128, false,  true, 1234ULL},
    {0.05f, -1.f, 1024, 256, false,  true, 1234ULL}
};

const std::vector<MeanCenterInputs<double> > inputsd = {
    {0.05,  1.0, 1024,  32,  true, false, 1234ULL},
    {0.05,  1.0, 1024,  64,  true, false, 1234ULL},
    {0.05,  1.0, 1024, 128,  true, false, 1234ULL},
    {0.05,  1.0, 1024, 256,  true, false, 1234ULL},
    {0.05, -1.0, 1024,  32, false, false, 1234ULL},
    {0.05, -1.0, 1024,  64, false, false, 1234ULL},
    {0.05, -1.0, 1024, 128, false, false, 1234ULL},
    {0.05, -1.0, 1024, 256, false, false, 1234ULL},
    {0.05,  1.0, 1024,  32,  true,  true, 1234ULL},
    {0.05,  1.0, 1024,  64,  true,  true, 1234ULL},
    {0.05,  1.0, 1024, 128,  true,  true, 1234ULL},
    {0.05,  1.0, 1024, 256,  true,  true, 1234ULL},
    {0.05, -1.0, 1024,  32, false,  true, 1234ULL},
    {0.05, -1.0, 1024,  64, false,  true, 1234ULL},
    {0.05, -1.0, 1024, 128, false,  true, 1234ULL},
    {0.05, -1.0, 1024, 256, false,  true, 1234ULL}
};

typedef MeanCenterTest<float> MeanCenterTestF;
TEST_P(MeanCenterTestF, Result) {
    ASSERT_TRUE(devArrMatch(0.f, mean_act, params.cols,
                            CompareApprox<float>(params.tolerance)));

    if (device_count > 1) {
    	ASSERT_TRUE(devArrMatch(0.f, mean_act_2, params.cols,
    	                    CompareApprox<float>(params.tolerance)));
    }
}

typedef MeanCenterTest<double> MeanCenterTestD;
TEST_P(MeanCenterTestD, Result){
    ASSERT_TRUE(devArrMatch(0.0, mean_act, params.cols,
                            CompareApprox<double>(params.tolerance)));

    if (device_count > 1) {
    	ASSERT_TRUE(devArrMatch(0.0, mean_act_2, params.cols,
    	                    CompareApprox<double>(params.tolerance)));
    }
}

INSTANTIATE_TEST_CASE_P(MeanCenterTests, MeanCenterTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(MeanCenterTests, MeanCenterTestD,
                        ::testing::ValuesIn(inputsd));

} // end namespace Stats
} // end namespace MLCommon
