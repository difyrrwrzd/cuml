#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "linalg/subtract.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename Type>
__global__ void naiveSubtractElemKernel(Type *out, const Type *in1,
                                        const Type *in2, int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = in1[idx] - in2[idx];
  }
}

template <typename Type>
void naiveSubtractElem(Type *out, const Type *in1, const Type *in2, int len) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naiveSubtractElemKernel<Type><<<nblks, TPB>>>(out, in1, in2, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename Type>
__global__ void naiveSubtractScalarKernel(Type *out, const Type *in1,
                                          const Type in2, int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = in1[idx] - in2;
  }
}

template <typename Type>
void naiveSubtractScalar(Type *out, const Type *in1, const Type in2, int len) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naiveSubtractScalarKernel<Type><<<nblks, TPB>>>(out, in1, in2, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct SubtractInputs {
  T tolerance;
  int len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const SubtractInputs<T> &dims) {
  return os;
}

template <typename T>
class SubtractTest : public ::testing::TestWithParam<SubtractInputs<T>> {
protected:
  void SetUp() override {
    params = ::testing::TestWithParam<SubtractInputs<T>>::GetParam();
    Random::Rng<T> r(params.seed);
    int len = params.len;
    allocate(in1, len);
    allocate(in2, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in1, len, T(-1.0), T(1.0));
    r.uniform(in2, len, T(-1.0), T(1.0));

    naiveSubtractElem(out_ref, in1, in2, len);
    naiveSubtractScalar(out_ref, out_ref, T(1), len);

    subtract(out, in1, in2, len);
    subtractScalar(out, out, T(1), len);
    subtract(in1, in1, in2, len);
    subtractScalar(in1, in1, T(1), len);

    CUDA_CHECK(hipGetDeviceCount(&device_count));
    if (device_count > 1) {
      T *h_in1 = (T *)malloc(len * sizeof(T));
      T *h_in2 = (T *)malloc(len * sizeof(T));
      updateHost(h_in1, in1, len);
      updateHost(h_in2, in2, len);
      subtractMGColSplitTest(h_in1, h_in2);
      free(h_in1);
      free(h_in2);
    }
  }

  void subtractMGColSplitTest(T *h_in1, T *h_in2) {
    int n_gpus = 2;

    TypeMG<T> d_in1[n_gpus];
    TypeMG<T> d_in2[n_gpus];
    TypeMG<T> d_out[n_gpus];

    for (int i = 0; i < n_gpus; i++) {
      d_in1[i].gpu_id = i;
      d_in2[i].gpu_id = i;
      d_out[i].gpu_id = i;
      CUDA_CHECK(hipSetDevice(d_in1[i].gpu_id));
      CUDA_CHECK(hipStreamCreate(&(d_in1[i].stream)));
      d_in2[i].stream = d_in1[i].stream;
      d_out[i].stream = d_in1[i].stream;
    }

    int len = params.len;
    allocateMG(d_in1, n_gpus, 1, len, true, true, false);
    allocateMG(d_in2, n_gpus, 1, len, true, true, false);
    allocateMG(d_out, n_gpus, 1, len, true, true, false);

    updateDeviceMG(d_in1, h_in1, n_gpus, false);
    updateDeviceMG(d_in1, h_in1, n_gpus, false);

    subtractMG(d_out, d_in1, d_in2, len, n_gpus, false);
    // subtractScalarMG(d_out, d_out, T(1), len, n_gpus, false);

    T *h_out = (T *)malloc(len * sizeof(T));
    updateHostMG(h_out, d_out, n_gpus, false);

    streamSyncMG(d_out, n_gpus);
    streamDestroyGPUs(d_out, n_gpus);

    freeMG(d_in1, n_gpus);
    freeMG(d_in2, n_gpus);
    freeMG(d_out, n_gpus);

    allocate(out_2, len);
    updateDevice(out_2, h_out, len);

    free(h_out);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));

    if (device_count > 1) {
      CUDA_CHECK(hipFree(out_2));
    }
  }

protected:
  SubtractInputs<T> params;
  T *in1, *in2, *out_ref, *out, *out_2;
  int device_count = 0;
};

const std::vector<SubtractInputs<float>> inputsf2 = {
  {0.000001f, 1024 * 1024, 1234ULL}};

const std::vector<SubtractInputs<double>> inputsd2 = {
  {0.00000001, 1024 * 1024, 1234ULL}};

typedef SubtractTest<float> SubtractTestF;
TEST_P(SubtractTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
                          CompareApprox<float>(params.tolerance)));

  if (device_count > 1) {
    ASSERT_TRUE(devArrMatch(out_ref, out_2, params.len,
                            CompareApprox<float>(params.tolerance)));
  }
}

typedef SubtractTest<double> SubtractTestD;
TEST_P(SubtractTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
                          CompareApprox<double>(params.tolerance)));

  if (device_count > 1) {
    ASSERT_TRUE(devArrMatch(out_ref, out_2, params.len,
                            CompareApprox<double>(params.tolerance)));
  }
}

INSTANTIATE_TEST_CASE_P(SubtractTests, SubtractTestF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(SubtractTests, SubtractTestD,
                        ::testing::ValuesIn(inputsd2));

} // end namespace LinAlg
} // end namespace MLCommon
