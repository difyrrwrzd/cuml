/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "linalg/strided_reduction.h"
#include "linalg/unary_op.h"
#include "random/rng.h"
#include "test_utils.h"

#include <thrust/device_vector.h>
#include <hipblas.h>

namespace MLCommon {
namespace LinAlg {

template <typename T>
struct stridedReductionInputs {
    T tolerance;
    int rows, cols;
    unsigned long long int seed;
};

template <typename T>
void stridedReductionLaunch(T *dots, const T *data, int cols, int rows,
                              hipStream_t stream) {
  stridedReduction(dots, data, cols, rows, (T)0, stream, false,
                   [] __device__(T in, int i) { return in * in; });
}


template <typename T, typename GEMV_t>
void unaryAndGemv(T *dots, const T *data, int cols, int rows, GEMV_t gemv,
                    hipStream_t stream){
    //computes a MLCommon unary op on data (squares it), then computes Ax
    //(A input matrix and x column vector) to sum columns
    thrust::device_vector<T> sq(cols*rows);
    unaryOp(thrust::raw_pointer_cast(sq.data()), data, cols*rows,
            [] __device__(T v) { return v*v; },
            stream);

    hipblasHandle_t handle;
    ASSERT_TRUE(hipblasCreate(&handle) == HIPBLAS_STATUS_SUCCESS);

    thrust::device_vector<T> ones(rows, 1); //column vector [1...1]
    T alpha = 1, beta = 0;
    ASSERT_TRUE(gemv(handle, HIPBLAS_OP_N, cols, rows,
                &alpha, thrust::raw_pointer_cast(sq.data()), cols,
                thrust::raw_pointer_cast(ones.data()), 1, &beta, 
                dots, 1) == HIPBLAS_STATUS_SUCCESS);
}

void unaryAndGemv(float *dots, const float *data, int cols, int rows,
                    hipStream_t stream){
    unaryAndGemv(dots, data, cols, rows, hipblasSgemv, stream);
}

void unaryAndGemv(double *dots, const double *data, int cols, int rows,
                    hipStream_t stream){
    unaryAndGemv(dots, data, cols, rows, hipblasDgemv, stream);
}


template <typename T>
class stridedReductionTest : public ::testing::TestWithParam<stridedReductionInputs<T>> {
protected:
  void SetUp() override {
    params = ::testing::TestWithParam<stridedReductionInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows*cols;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    allocate(data, len);
    allocate(dots_exp, cols); //expected dot products (from test)
    allocate(dots_act, cols); //actual dot products (from prim)
    r.uniform(data, len, T(-1.0), T(1.0), stream); //initialize matrix to random

    unaryAndGemv(dots_exp, data, cols, rows, stream);
    stridedReductionLaunch(dots_act, data, cols, rows, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(dots_exp));
    CUDA_CHECK(hipFree(dots_act));
  }

protected:
  stridedReductionInputs<T> params;
  T *data, *dots_exp, *dots_act;
};


const std::vector<stridedReductionInputs<float>> inputsf = {
  {0.00001f, 1024,  32, 1234ULL},
  {0.00001f, 1024,  64, 1234ULL},
  {0.00001f, 1024, 128, 1234ULL},
  {0.00001f, 1024, 256, 1234ULL},
  {0.00001f, 1024,  32, 1234ULL},
  {0.00001f, 1024,  64, 1234ULL},
  {0.00001f, 1024, 128, 1234ULL},
  {0.00001f, 1024, 256, 1234ULL}
};

const std::vector<stridedReductionInputs<double>> inputsd = {
  {0.000000001, 1024,  32, 1234ULL},
  {0.000000001, 1024,  64, 1234ULL},
  {0.000000001, 1024, 128, 1234ULL},
  {0.000000001, 1024, 256, 1234ULL},
  {0.000000001, 1024,  32, 1234ULL},
  {0.000000001, 1024,  64, 1234ULL},
  {0.000000001, 1024, 128, 1234ULL},
  {0.000000001, 1024, 256, 1234ULL}
};

typedef stridedReductionTest<float> stridedReductionTestF;
TEST_P(stridedReductionTestF, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.cols,
                          CompareApprox<float>(params.tolerance)));
}

typedef stridedReductionTest<double> stridedReductionTestD;
TEST_P(stridedReductionTestD, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.cols,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(stridedReductionTests, stridedReductionTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(stridedReductionTests, stridedReductionTestD, ::testing::ValuesIn(inputsd));

} // end namespace LinAlg
} // end namespace MLCommon
