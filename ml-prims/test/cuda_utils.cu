#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "cuda_utils.h"


namespace MLCommon {

TEST(Utils, Assert) {
    ASSERT_NO_THROW(ASSERT(1 == 1, "Should not assert!"));
    ASSERT_THROW(ASSERT(1 != 1, "Should assert!"), std::runtime_error);
}

TEST(Utils, CudaCheck) {
    ASSERT_NO_THROW(CUDA_CHECK(hipFree(nullptr)));
}

// we want the functions like 'log2' to work both at compile and runtimes!
static const int log2Of1024 = log2(1024);
static const int log2Of1023 = log2(1023);
TEST(Utils, log2) {
    ASSERT_EQ(10, log2(1024));
    ASSERT_EQ(9, log2(1023));
    ASSERT_EQ(10, log2Of1024);
    ASSERT_EQ(9, log2Of1023);
}

} // end namespace MLCommon
