/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "csr.h"
#include <gtest/gtest.h>
#include "sparse/csr.h"
#include "random/rng.h"
#include "test_utils.h"

#include <iostream>

namespace MLCommon {
namespace Sparse {

template <typename T>
class CSRTest : public ::testing::TestWithParam<CSRInputs<T>> {
protected:
  void SetUp() override {}

  void TearDown() override {}

protected:
  CSRInputs<T> params;
};

const std::vector<CSRInputs<float>> inputsf = {
  {5, 10, 5, 1234ULL}};

typedef CSRTest<float> CSRRowNormalizeL1;
TEST_P(CSRRowNormalizeL1, Result) {

    int *ex_scan;
    float *in_vals, *result, *verify;

    int ex_scan_h[4] = {0, 4, 8, 9 };
    float in_vals_h[10] = { 1.0, 1.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0 };

    float verify_h[10] =  { 0.5, 0.5, 0.0, 0.0, 0.5, 0.5, 0.0, 0.0, 1, 0.0 };

    allocate(in_vals, 10);
    allocate(verify, 10);
    allocate(ex_scan, 4);
    allocate(result, 10, true);

    updateDevice(ex_scan, *&ex_scan_h, 4);
    updateDevice(in_vals, *&in_vals_h, 10);
    updateDevice(verify, *&verify_h, 10);

    dim3 grid(ceildiv(10, 32), 1, 1);
    dim3 blk(32, 1, 1);

    csr_row_normalize_l1<32, float><<<grid, blk>>>(ex_scan, in_vals, 10, 4, result);
    hipDeviceSynchronize();

    ASSERT_TRUE(devArrMatch<float>(verify, result, 10, Compare<float>()));

    CUDA_CHECK(hipFree(ex_scan));
    CUDA_CHECK(hipFree(in_vals));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
}

typedef CSRTest<float> CSRSum;
TEST_P(CSRSum, Result) {

    int *ex_scan, *ind_ptr_a, *ind_ptr_b;
    float *in_vals_a, *in_vals_b, *result, *verify;

    int ex_scan_h[4] = {0, 4, 8, 9 };

    int indptr_a_h[10] = { 1, 2, 3, 4, 1, 2, 3, 5, 0, 1 };
    int indptr_b_h[10] = { 1, 2, 5, 4, 0, 2, 3, 5, 1, 0 };

    float in_vals_h[10] = { 1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 0.0 };

    float verify_h[10] =  { 2.0, 2.0, 0.5, 0.5, 0.2, 0.2, 0.0, 0.0, 0.2, 0.0 };

    allocate(in_vals_a, 10);
    allocate(in_vals_b, 10);
    allocate(verify, 10);
    allocate(ex_scan, 4);
    allocate(result, 10, true);

    allocate(ind_ptr_a, 10);
    allocate(ind_ptr_b, 10);

    updateDevice(ex_scan, *&ex_scan_h, 4);
    updateDevice(in_vals_a, *&in_vals_h, 10);
    updateDevice(in_vals_b, *&in_vals_h, 10);
    updateDevice(verify, *&verify_h, 10);
    updateDevice(ind_ptr_a, *&indptr_a_h, 10);
    updateDevice(ind_ptr_b, *&indptr_b_h, 10);

    int *result_ind;
    allocate(result_ind, 4);

    int nnz = 0;

    csr_add_calc_inds<float, 32>(
        ex_scan, ind_ptr_a, in_vals_a,
        ex_scan, ind_ptr_b, in_vals_b,
        10, 4,
        &nnz,
        result_ind
    );

    std::cout << MLCommon::arr2Str(result_ind, 4, "result_ind") << std::endl;
    std::cout << "final_nnz=" << nnz << std::endl;

    int *result_indptr;
    float *result_val;
    allocate(result_indptr, nnz);
    allocate(result_val, nnz);

    csr_add_finalize<float, 32>(
        ex_scan, ind_ptr_a, in_vals_a,
        ex_scan, ind_ptr_b, in_vals_b,
        10, 4,
        result_ind, result_indptr, result_val
    );
    std::cout << MLCommon::arr2Str(result_indptr, nnz, "result_intptr") << std::endl;

    std::cout << MLCommon::arr2Str(result_val, nnz, "result") << std::endl;

    ASSERT_TRUE(devArrMatch<float>(verify, result, 10, Compare<float>()));

    CUDA_CHECK(hipFree(ex_scan));
    CUDA_CHECK(hipFree(in_vals_a));
    CUDA_CHECK(hipFree(in_vals_b));
    CUDA_CHECK(hipFree(ind_ptr_a));
    CUDA_CHECK(hipFree(ind_ptr_b));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
    CUDA_CHECK(hipFree(result_indptr));
    CUDA_CHECK(hipFree(result_val));
}


INSTANTIATE_TEST_CASE_P(CSRTests, CSRRowNormalizeL1,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(CSRTests, CSRSum,
                        ::testing::ValuesIn(inputsf));
}}

