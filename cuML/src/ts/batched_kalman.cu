#include "hip/hip_runtime.h"
#include "kalman.h"
#include "batched_kalman.h"
#include <matrix/batched_matrix.h>
#include <utils.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/for_each.h>
#include <iostream>
#include <cstdio>

#include <fstream>
#include <unistd.h>

#include <nvToolsExt.h>

#include <hipcub/hipcub.hpp>

#include <chrono>
#include <ratio>

// #include <thrust/lo

using std::vector;

using MLCommon::Matrix::BatchedMatrix;
using MLCommon::Matrix::BatchedMatrixMemoryPool;
using MLCommon::Matrix::b_gemm;
using MLCommon::Matrix::b_kron;
using MLCommon::Matrix::b_solve;
using MLCommon::allocate;
using MLCommon::updateDevice;
using MLCommon::updateHost;

////////////////////////////////////////////////////////////
#include <iostream>

void nvtx_range_push(std::string msg) {
  nvtxRangePush(msg.c_str());
}

void nvtx_range_pop() {
  nvtxRangePop();
}

void process_mem_usage(double& vm_usage, double& resident_set)
{
  vm_usage     = 0.0;
  resident_set = 0.0;

  // the two fields we want
  unsigned long vsize;
  long rss;
  {
    std::string ignore;
    std::ifstream ifs("/proc/self/stat", std::ios_base::in);
    ifs >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore
        >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore >> ignore
        >> ignore >> ignore >> vsize >> rss;
  }

  long page_size_kb = sysconf(_SC_PAGE_SIZE) / 1024; // in case x86-64 is configured to use 2MB pages
  vm_usage = vsize / 1024.0;
  resident_set = rss * page_size_kb;
}
////////////////////////////////////////////////////////////


// __global__ void vs_eq_ys_m_alpha00_kernel(double* d_vs, int it,
//                                           const double* ys_it,
//                                           double** alpha, int r,
//                                           int num_batches) {
//   int batch_id = blockIdx.x*blockDim.x + threadIdx.x;
//   if(batch_id < num_batches) {
//     d_vs[it*num_batches + batch_id] = ys_it[batch_id] - alpha[batch_id][0];
//   }
// }

// void vs_eq_ys_m_alpha00(double* d_vs,int it,const vector<double*>& ptr_ys_b,const BatchedMatrix& alpha) {
//   const int num_batches = alpha.batches();
//   const int block_size = 16;
//   const int num_blocks = std::ceil((double)num_batches/(double)block_size);

//   vs_eq_ys_m_alpha00_kernel<<<num_blocks, block_size>>>(d_vs, it, ptr_ys_b[it],
//                                                         alpha.data(), alpha.shape().first, num_batches);
//   CUDA_CHECK(hipPeekAtLastError());
  
// }

// __global__ void fs_it_P00_kernel(double* d_Fs, int it, double** P, int num_batches) {
//   int batch_id = blockIdx.x*blockDim.x + threadIdx.x;
//   if(batch_id < num_batches) {
//     d_Fs[it*num_batches + batch_id] = P[batch_id][0];
//   }
// }

// void fs_it_P00(double* d_Fs, int it, const BatchedMatrix& P) {

//   const int block_size = 16;
//   const int num_batches = P.batches();
//   const int num_blocks = std::ceil((double)num_batches/(double)block_size);

//   fs_it_P00_kernel<<<num_blocks, block_size>>>(d_Fs, it, P.data(), num_batches);
//   CUDA_CHECK(hipPeekAtLastError());

// }

// __global__ void _1_Fsit_TPZt_kernel(double* d_Fs, int it, double** TPZt,
//                                     int N_TPZt, // size of matrix TPZt
//                                     int num_batches,
//                                     double** K // output
//                                     ) {
  
//   int batch_id = blockIdx.x;
//   for(int i=0;i<N_TPZt/blockDim.x;i++) {
//     int ij = threadIdx.x + i*blockDim.x;
//     if(ij < N_TPZt) {
//       K[batch_id][ij] = 1.0/d_Fs[batch_id + num_batches * it] * TPZt[batch_id][ij];
//     }
//   }
// }

// BatchedMatrix _1_Fsit_TPZt(double* d_Fs, int it, const BatchedMatrix& TPZt) {
//   BatchedMatrix K(TPZt.shape().first, TPZt.shape().second, TPZt.batches());

//   const int TPZt_size = TPZt.shape().first * TPZt.shape().second;
//   const int block_size = (TPZt_size) % 128;
  
//   const int num_batches = TPZt.batches();
//   const int num_blocks = num_batches;

//   // call kernel
//   _1_Fsit_TPZt_kernel<<<num_blocks,block_size>>>(d_Fs, it, TPZt.data(), TPZt_size, num_batches, K.data());
//   CUDA_CHECK(hipPeekAtLastError());

//   return K;
// }

BatchedMatrix Kvs_it(const BatchedMatrix& K, double* d_vs, int it) {
  BatchedMatrix Kvs(K.shape().first, K.shape().second, K.batches(), K.pool());
  auto num_batches = K.batches();
  auto counting = thrust::make_counting_iterator(0);
  double** d_K = K.data();
  double** d_Kvs = Kvs.data();
  int m = K.shape().first;
  int n = K.shape().second;
  thrust::for_each(counting, counting + num_batches,
                   [=]__device__(int bid) {
                     double vs = d_vs[bid + it*num_batches];
                     for(int ij=0; ij<m*n; ij++) {
                       d_Kvs[bid][ij] = d_K[bid][ij]*vs;
                     }
                   });
  return Kvs;
}

__global__ void sumLogFs_kernel(double* d_Fs, int num_batches, int nobs, double* d_sumLogFs) {
  double sum = 0.0;
  int bid = threadIdx.x + blockIdx.x*blockDim.x;
  if(bid < num_batches) {
    for (int it = 0; it < nobs; it++) {
      sum += log(d_Fs[bid]);
    }
    d_sumLogFs[bid] = sum;
  }
}

double* sumLogFs(double* d_Fs, const int num_batches, const int nobs) {

  double* d_sumLogFs;
  allocate(d_sumLogFs, num_batches);
  // compute sum(log(Fs[0:nobs]))
  // const int block_size = 32;
  // const int num_blocks = std::ceil((double)num_batches/(double)block_size);
  // sumLogFs_kernel<<<num_blocks, block_size>>>(d_Fs, num_batches, nobs, d_sumLogFs);
  // CUDA_CHECK(hipPeekAtLastError());
  auto counting = thrust::make_counting_iterator(0);
  thrust::for_each(counting, counting + num_batches,
                   [=]__device__(int bid) {
                     double sum = 0.0;
                     for (int it = 0; it < nobs; it++) {
                       sum += log(d_Fs[bid + it*num_batches]);
                     }
                     d_sumLogFs[bid] = sum;
                   });
  CUDA_CHECK(hipPeekAtLastError());
  return d_sumLogFs;
}


void batched_kalman_filter_cpu(const vector<double*>& h_ys_b, // { vector size batches, each item size nobs }
                               int nobs,
                               const vector<double*>& h_Zb, // { vector size batches, each item size Zb }
                               const vector<double*>& h_Rb, // { vector size batches, each item size Rb }
                               const vector<double*>& h_Tb, // { vector size batches, each item size Tb }
                               int r,
                               vector<double>& h_loglike_b,
                               vector<vector<double>>& h_vs_b,
                               bool initP_with_kalman_iterations
                               ) {

  nvtxRangePush(__FUNCTION__);

  const size_t num_batches = h_Zb.size();
  
  h_loglike_b.resize(num_batches);
  h_vs_b.resize(num_batches);
  for(int i=0; i<num_batches; i++) {
    h_vs_b[i].resize(nobs);
  }
  for(int bi=0; bi<num_batches; bi++) {
    kalman_filter(h_ys_b[bi], nobs,
                  h_Zb[bi], h_Rb[bi], h_Tb[bi],
                  r,
                  h_vs_b[bi].data(),
                  &h_loglike_b[bi],
                  initP_with_kalman_iterations
                  );
  }
  
  nvtxRangePop();
}

__device__ void Mv(double* A, double* v, int r, int tid, double* out) {
  out[tid] = 0.0;
  if(tid < r) {
    for(int i=0; i<r; i++) {
      out[tid] += A[tid + r*i] * v[i];
    }
  }
}

__device__ void MM(double *A, double *B, int r, int tid, double *out) {

  out[tid] = 0.0;
  for(int i=0; i<r; i++) {
    
    // access pattern should be:
    // out[0] += A[0 + r*i] * B[i + 0*r];
    // out[1] += A[1 + r*i] * B[i + 0*r];
    // out[2] += A[0 + r*i] * B[i + 1*r];
    // out[3] += A[1 + r*i] * B[i + 1*r];
    
    out[tid] += A[tid%r + r*i]*B[i + (tid/r % r) *r];
  }

}

extern __shared__ double s_array[]; // size = r*r x 5 + r x 3
__global__ void batched_kalman_loop_kernel(double* ys, int nobs,
                                           double** T, // \in R^(r x r)
                                           double** Z, // \in R^(1 x r)
                                           double** RRT, // \in R^(r x r)
                                           double** P, // \in R^(r x r)
                                           double** alpha, // \in R^(r x 1)
                                           int r,
                                           int num_batches,
                                           double* vs,
                                           double* Fs,
                                           double* sum_logFs
                                           ) {

  // kalman matrices and temporary storage
  int r2 = r*r;
  double* s_RRT = &s_array[0]; // rxr
  double* s_T = &s_array[r2]; // rxr
  double* s_Z = &s_array[2*r2]; // r
  double* s_P = &s_array[2*r2+r]; // rxr
  double* s_alpha = &s_array[3*r2+r]; // r
  double* s_K = &s_array[3*r2+2*r]; // r
  double* tmpA = &s_array[3*r2+3*r]; // rxr
  double* tmpB = &s_array[4*r2+3*r]; // rxr

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;

  // preload kalman matrices from GM.
  s_RRT[tid] = RRT[bid][tid];
  s_T[tid] = T[bid][tid];
  s_P[tid] = P[bid][tid];
  if(tid < r) {
    s_Z[tid] = Z[bid][tid];
    s_alpha[tid]= alpha[bid][tid];
  }
  __syncthreads();

  double bid_sum_logFs = 0.0;

  for(int it=0; it<nobs; it++) {

    // 1. & 2.
    // vs[it] = ys[it] - alpha(0,0);
    // Fs[it] = P(0,0);
    if(tid==0) {
      vs[it + bid*nobs] = ys[it + bid*nobs] - s_alpha[0];
      Fs[it + bid*nobs] = s_P[0];
      bid_sum_logFs += log(s_P[0]);
    }
    __syncthreads();
  
    // 3.
    // MatrixT K = 1.0/Fs[it] * (T * P * Z.transpose());
    // tmpA = P*Z.T
    Mv(s_P, s_Z, r, tid, tmpA);
    __syncthreads();
    // tmpB = T*tmpA
    Mv(s_T, tmpA, r, tid, tmpB);
    __syncthreads();
    // tmpB = 1/Fs[it] * tmpB
    if(tid < r) {
      s_K[tid] = 1/Fs[it + bid*nobs] * tmpB[tid];
    }
    __syncthreads();
  

    // 4.
    // alpha = T*alpha + K*vs[it];
    Mv(s_T, s_alpha, r, tid, tmpA);
    if (tid < r) {
      s_alpha[tid] = tmpA[tid] + s_K[tid] * vs[it + bid * nobs];
    }
    __syncthreads();

    // 5.
    // MatrixT L = T - K*Z;
    // tmpA = KZ
    // tmpA[0] = K[0]*Z[0]
    // tmpA[1] = K[1]*Z[0]
    // tmpA[2] = K[0]*Z[1]
    // tmpA[3] = K[1]*Z[1]
    // pytest [i % 3 for i in range(9)] -> 0 1 2 0 1 2 0 1 2
    // pytest [i//3 % 3 for i in range(9)] -> 0 0 0 1 1 1 2 2 2

    tmpA[tid] = s_K[tid % r] * s_Z[(tid / r) % r];

    __syncthreads();
    // tmpA = T-tmpA
    tmpA[tid] = s_T[tid] - tmpA[tid];
    __syncthreads();
    // L = tmpA

    // 6.
    // tmpB = tmpA.transpose()
    tmpB[tid] = tmpA[tid * r + tid / r % r];
    // L.T = tmpB
    __syncthreads();

    // P = T * P * L.transpose() + R * R.transpose();
    // tmpA = P*L.T
    MM(s_P, tmpB, r, tid, tmpA);
    __syncthreads();
    // tmpB = T*tmpA;
    MM(s_T, tmpA, r, tid, tmpB);
    __syncthreads();
    // P = tmpB + RRT
    s_P[tid] = tmpB[tid] + s_RRT[tid];
    __syncthreads();
  }
  if(tid == 0) {
    sum_logFs[bid] = bid_sum_logFs;
  }
}

void batched_kalman_loop(double* ys, int nobs,
                         const BatchedMatrix& T,
                         const BatchedMatrix& Z,
                         const BatchedMatrix& RRT,
                         const BatchedMatrix& P0,
                         const BatchedMatrix& alpha,
                         int r,
                         double* vs,
                         double* Fs,
                         double* sum_logFs
                         ) {

  const int num_batches = T.batches();
  const int num_blocks = num_batches;
  const int num_threads = r*r;
  const size_t bytes_shared_memory = (5*r*r + 3*r) * sizeof(double);
  
  batched_kalman_loop_kernel<<<num_blocks, num_threads, bytes_shared_memory>>>(ys, nobs,
                                                                               T.data(), Z.data(),
                                                                               RRT.data(), P0.data(),
                                                                               alpha.data(),
                                                                               r,
                                                                               num_batches,
                                                                               vs, Fs,
                                                                               sum_logFs
                                                                               );

  CUDA_CHECK(hipPeekAtLastError());
  CUDA_CHECK(hipDeviceSynchronize());

}

__global__ void batched_kalman_loglike_kernel(double *d_vs, double *d_Fs, double *d_sumLogFs,
                               int nobs, int num_batches, double *sigma2,
                               double *loglike) {

  using BlockReduce = hipcub::BlockReduce<double, 128>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int num_threads = blockDim.x;
  double bid_sigma2 = 0.0;
  for(int it=0; it<nobs; it+=num_threads) {
    // vs and Fs are in time-major order
    int idx = (it + tid) + bid * nobs;
    double d_vs2_Fs = 0.0;
    if (idx < nobs*num_batches) {
      d_vs2_Fs = d_vs[idx] * d_vs[idx] / d_Fs[idx];
    }
    __syncthreads();
    double partial_sum = BlockReduce(temp_storage).Sum(d_vs2_Fs, nobs - it);
    bid_sigma2 += partial_sum;
  }
  if(tid == 0) {
    bid_sigma2 /= nobs;
    sigma2[bid] = bid_sigma2;
    loglike[bid] = -.5 * (d_sumLogFs[bid] + nobs * log(bid_sigma2)) - nobs / 2. * (log(2 * M_PI) + 1);
  }
}

void batched_kalman_loglike(double* d_vs, double* d_Fs, double* d_sumLogFs, int nobs, int num_batches,
                    double* sigma2, double* loglike) {

  // BlockReduce uses 128 threads, so here also use 128 threads.
  const int num_threads = 128;
  batched_kalman_loglike_kernel<<<num_batches, num_threads>>>(d_vs, d_Fs, d_sumLogFs, nobs, num_batches,
                                                              sigma2, loglike);
  CUDA_CHECK(hipDeviceSynchronize());

}

// Internal Kalman filter implementation that assumes data exists on GPU.
void _batched_kalman_filter(double* d_ys,
                            int nobs,
                            const BatchedMatrix& Zb,
                            const BatchedMatrix& Tb,
                            const BatchedMatrix& Rb,
                            int r,
                            double* d_vs,
                            double* d_Fs,
                            double* d_loglike,
                            double* d_sigma2,
                            bool initP_with_kalman_iterations=true) {

  const size_t num_batches = Zb.batches();

  BatchedMatrix RRT = b_gemm(Rb, Rb, false, true);
  
  BatchedMatrix P(r, r, num_batches, Zb.pool(), false);
  if(initP_with_kalman_iterations)
    // A single Kalman iteration
    P = b_gemm(Tb,Tb,false,true) - Tb * b_gemm(Zb,b_gemm(Zb,Tb,false,true),true,false) + RRT;
  else {
    // # (Durbin Koopman "Time Series Analysis" pg 138)
    // NumPy version
    //   invImTT = np.linalg.pinv(np.eye(r**2) - np.kron(T_bi, T_bi))
    //   P0 = np.reshape(invImTT @ (R_bi @ R_bi.T).ravel(), (r, r), order="F")
    BatchedMatrix I_m_TxT = BatchedMatrix::Identity(r*r, num_batches, Zb.pool()) - b_kron(Tb, Tb);
    BatchedMatrix invI_m_TxT_x_RRTvec = b_solve(I_m_TxT, RRT.vec());
    BatchedMatrix P0 = invI_m_TxT_x_RRTvec.mat(r, r);
    P = P0;
    // auto& stream = std::cout;
    // stream.precision(16);
    // MLCommon::myPrintDevVector("P0", P[0], 4*P0.batches(), stream);
  }

  // init alpha to zero
  BatchedMatrix alpha(r, 1, num_batches, Zb.pool(), true);

  // init vs, Fs
  // In batch-major format.
  double* d_sumlogFs;
  
  allocate(d_sumlogFs, num_batches);

  CUDA_CHECK(hipPeekAtLastError());
  
  // Reference implementation
  // For it = 1:nobs
  //  // 1.
  //   vs[it] = ys[it] - alpha(0,0);
  //  // 2.
  //   Fs[it] = P(0,0);

  //   if(Fs[it] < 0) {
  //     std::cout << "P=" << P << "\n";
  //     throw std::runtime_error("ERROR: F < 0");
  //   }
  //   3.
  //   MatrixT K = 1.0/Fs[it] * (T * P * Z.transpose());
  //   4.
  //   alpha = T*alpha + K*vs[it];
  //   5.
  //   MatrixT L = T - K*Z;
  //   6.
  //   P = T * P * L.transpose() + R * R.transpose();
  //   loglikelihood += std::log(Fs[it]);
  // }

  batched_kalman_loop(d_ys, nobs, Tb, Zb, RRT, P, alpha, r, d_vs, d_Fs, d_sumlogFs);

  // Finalize loglikelihood
  // 7. & 8.
  // double sigma2 = ((vs.array().pow(2.0)).array() / Fs.array()).mean();
  // double loglike = -.5 * (loglikelihood + nobs * std::log(sigma2));
  // loglike -= nobs / 2. * (std::log(2 * M_PI) + 1);
  
  batched_kalman_loglike(d_vs, d_Fs, d_sumlogFs, nobs, num_batches, d_sigma2, d_loglike);

  
}

void batched_kalman_filter(double* h_ys,
                           int nobs,
                           const vector<double*>& h_Zb, // { vector size batches, each item size Zb }
                           const vector<double*>& h_Rb, // { vector size batches, each item size Rb }
                           const vector<double*>& h_Tb, // { vector size batches, each item size Tb }
                           int r,
                           int num_batches,
                           std::vector<double>& h_loglike_b,
                           std::vector<vector<double>>& h_vs_b,
                           bool initP_with_kalman_iterations
                           ) {

  nvtxRangePush(__FUNCTION__);

  const size_t ys_len = nobs;

  ////////////////////////////////////////////////////////////
  // xfer from host to device
  double* d_ys;
  allocate(d_ys, nobs*num_batches);
  updateDevice(d_ys, h_ys, nobs*num_batches, 0);

  auto memory_pool = std::make_shared<BatchedMatrixMemoryPool>(num_batches);

  BatchedMatrix Zb(1, r, num_batches, memory_pool);
  BatchedMatrix Tb(r, r, num_batches, memory_pool);
  BatchedMatrix Rb(r, 1, num_batches, memory_pool);

  ////////////////////////////////////////////////////////////
  // Copy matrices to device
  {
    //Tb
    std::vector<double> matrix_copy(r*r*num_batches);
    for(int bi=0;bi<num_batches;bi++) {
      for(int i=0;i<r*r;i++) {
        matrix_copy[i + bi*r*r] = h_Tb[bi][i];
      }
    }
    updateDevice(Tb[0],matrix_copy.data(),r*r*num_batches, 0);

    //Zb
    for(int bi=0;bi<num_batches;bi++) {
      for(int i=0;i<r;i++) {
        matrix_copy[i + bi*r] = h_Zb[bi][i];
      }
    }
    updateDevice(Zb[0],matrix_copy.data(),r*num_batches, 0);

    // Rb
    for(int bi=0;bi<num_batches;bi++) {
      for(int i=0;i<r;i++) {
        matrix_copy[i + bi*r] = h_Rb[bi][i];
      }
    }
    updateDevice(Rb[0],matrix_copy.data(),r*num_batches, 0);
  }


  ////////////////////////////////////////////////////////////
  // Computation
  double* d_vs; // time-major order
  double* d_Fs; // time-major order
  allocate(d_vs, ys_len*num_batches);
  allocate(d_Fs, ys_len*num_batches);
  
  double* d_loglike;
  double* d_sigma2;
  allocate(d_sigma2, num_batches);
  allocate(d_loglike, num_batches);

  _batched_kalman_filter(d_ys, nobs, Zb, Tb, Rb, r, d_vs, d_Fs, d_loglike, d_sigma2,
                         initP_with_kalman_iterations);

  ////////////////////////////////////////////////////////////
  // xfer results from GPU
  h_loglike_b.resize(num_batches);
  updateHost(h_loglike_b.data(), d_loglike, num_batches, 0);

  vector<double> h_vs(ys_len*num_batches);
  updateHost(h_vs.data(), d_vs, ys_len*num_batches, 0);

  h_vs_b.resize(num_batches);
  for(int i=0;i<num_batches;i++) {
    h_vs_b[i].resize(ys_len);
    for(int j=0;j<ys_len;j++) {
      h_vs_b[i][j] = h_vs[j + i*ys_len]; // vs is in time-major order
    }
  }

  ////////////////////////////////////////////////////////////
  // free memory
  CUDA_CHECK(hipFree(d_vs));
  CUDA_CHECK(hipFree(d_Fs));
  CUDA_CHECK(hipFree(d_sigma2));
  CUDA_CHECK(hipFree(d_loglike));
  nvtxRangePop();
  
}
