//This file is Vishal's personal iris test as a checker, and decision tree verifier

#include <stdio.h>
#include <iostream>
#include "tree.cuh"
#include <fstream>
#include <sstream>
#include <string>

#define N 150

using namespace std;
int main()
{
  ifstream myfile;
  myfile.open("data.csv");
  string line;
  vector<float> data;
  vector<int> labels;
  int counter = 0;
  data.resize(N*4);
  labels.resize(N);
  
  while(getline(myfile,line))
    {
      stringstream str(line);
      vector<float> row;
      float i;
      while ( str >> i)
	{
	  row.push_back(i);
	  if(str.peek() == ',')
	    str.ignore();
	}
      data[counter + 0*N] = row[0];
      data[counter + 1*N] = row[1];
      data[counter + 2*N] = row[2];
      data[counter + 3*N] = row[3];
      labels[counter] = (int)row[4];
      counter++;
    }
  cout << "Lines processed " << counter << endl;  
  myfile.close();

  float *d_data;
  int *d_labels;
  
  hipMalloc((void**)(&d_data),N*4*sizeof(float));
  hipMalloc((void**)(&d_labels),N*sizeof(int));

  hipMemcpy(d_data,data.data(),N*4*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_labels,labels.data(),N*sizeof(int),hipMemcpyHostToDevice);
    
  ML::DecisionTree::DecisionTreeClassifier model;
  model.plant(d_data,4,N,1.0,d_labels);

	      
  hipFree(d_data);
  hipFree(d_labels);
  return 0;
  
}
