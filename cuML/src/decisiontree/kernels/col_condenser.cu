/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#pragma once
#include "hipcub/hipcub.hpp"

/* 

Assumptions: 
	- There will be a 64-bit mask (row_mask)  associated with each row of a dataset per tree. This value can be changed atomically (as different nodes in that same tree are built in //).
    - A 64-bit mask will allow us to go up to 64-levels deep which seems to be sufficient (good enough)  according to our SKL-rf-def experiments.

Inputs:
	- data points to entire dataset in col major format.
	- cur_tree_depth: an int that tells us to look at the least significant (cur_tree_depth - 1) bits of the row_mask. 
		=> So, if we're about to build a child of the root, we'll consider all rows in the bootstrapped sample.
		=> If we're about to build a child of the left child of the root, we'll consider all rows with the least significant bit set to 0. 
	- node_mask: the identifier of the parent node. We only care for rows where the first (cur_tree_depth - 1) bits of the node match with this node_mask identifier
		
	- n_rows the original number of rows in the dataset.
	- col: the column we care about


Output:
	- condensed column to look up that includes only the relevant rows. 
	- condensed labels 

*/

using namespace hipcub;

//Dummy Operator. Always return true. It's the transformation we care about.
struct Match
{
    __host__ __device__ __forceinline__ Match() {}

    __host__ __device__ __forceinline__
    bool operator()(const unsigned long long &a) const {
		return true;
	}
};


//Return true when the mask of a row matches the node mask for a given depth. 
struct RowMatch {
	unsigned long long node_mask;
	int depth;
	
    __host__ __device__ __forceinline__
    RowMatch(unsigned long long node_mask, int depth) : node_mask(node_mask), depth(depth) {}

	__host__ __device__ __forceinline__
	bool operator()(const unsigned long long & row_mask) const {
		return (((row_mask ^ node_mask) & ((1 << depth)-1)) == 0);
	}	
};


void col_condenser(float * input_data, int * labels, unsigned long long * row_masks, int col_id, 
				   const int n_rows, const int n_cols, int cur_tree_depth, 
				   unsigned long long node_mask, float * condensed_col, int * condensed_labels) {

	/* First step: generate flags memory array using a TransformInputIterator.
	   All rows will be selected, and the flags elements will only be set for the rows that matter. 
	*/

	bool * flags;
	int * n_selected_rows;
	hipMalloc((void**)&flags, n_rows * sizeof(bool));
	hipMalloc((void**)&condensed_col, n_rows * sizeof(float));
	hipMalloc((void**)&condensed_labels, n_rows * sizeof(int));
	hipMalloc((void**)&n_selected_rows, sizeof(int));
	
	Match select_op;
	RowMatch conversion_op(node_mask, cur_tree_depth); 

	// Assumption: row_masks is a device pointer. 
	hipcub::TransformInputIterator<bool, RowMatch, unsigned long long *> itr(row_masks, conversion_op);

    void * tmp_storage = NULL;
    size_t tmp_storage_bytes = 0;

    CubDebugExit(DeviceSelect::If(tmp_storage, tmp_storage_bytes, itr, flags, n_selected_rows, n_rows, select_op)); //n_selected_rows will be n_rows

	hipMalloc(&tmp_storage, tmp_storage_bytes);

    CubDebugExit(DeviceSelect::If(tmp_storage, tmp_storage_bytes, itr,
                 flags, n_selected_rows, n_rows, select_op));
	hipFree(tmp_storage);


	/* Second step: use the previously generated flags array to condense the col_id column.
	   of input_data.
	*/

	tmp_storage = NULL;
	tmp_storage_bytes = 0;


    CubDebugExit(DeviceSelect::Flagged(tmp_storage, tmp_storage_bytes, &input_data[col_id * n_rows], 
				flags, condensed_col, n_selected_rows, n_rows));

	hipMalloc(&tmp_storage, tmp_storage_bytes);

    CubDebugExit(DeviceSelect::Flagged(tmp_storage, tmp_storage_bytes, &input_data[col_id * n_rows], 
				flags, condensed_col, n_selected_rows, n_rows));




	/* Final step: Select labels */	

	tmp_storage = NULL;
	tmp_storage_bytes = 0;

    CubDebugExit(DeviceSelect::Flagged(tmp_storage, tmp_storage_bytes, labels,
				flags, condensed_labels, n_selected_rows, n_rows));

	hipMalloc(&tmp_storage, tmp_storage_bytes);

    CubDebugExit(DeviceSelect::Flagged(tmp_storage, tmp_storage_bytes, labels,
				flags,  condensed_labels, n_selected_rows, n_rows));



	// Cleanup
	hipFree(tmp_storage);
	hipFree(flags);
	hipFree(n_selected_rows);


}

