/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "svm/workingset.h"
#include "svm/smosolver.h"
#include <gtest/gtest.h>
#include <cuda_utils.h>
#include <test_utils.h>
#include <iostream>
#include <hipcub/hipcub.hpp>   // or equivalently <cub/device/device_radix_sort.cuh>

namespace ML {
namespace SVM {
using namespace MLCommon;


/*
template<typename math_t>
class SmoSolverTest: public ::testing::Test {
protected:
   // SmoSolver<math_t> * smo;
    SmoSolverTest() 
    {
     // smo = new SmoSolver<math_t>(10,4);
    }
    ~ SmoSolverTest() {
      //delete smo;
    }
};

typedef SmoSolverTest<float> SmoSolverTestF;
*/


TEST(SmoSolverTestF, SelectWorkingSetTest) {
  WorkingSet<float> *ws;
  
  ws = new WorkingSet<float>(10);
  EXPECT_EQ(ws->GetSize(), 10);
  delete ws;
  
  ws = new WorkingSet<float>(100000);
  EXPECT_EQ(ws->GetSize(), 1024);
  delete ws;

  ws = new WorkingSet<float>(10, 4);
  EXPECT_EQ(ws->GetSize(), 4);
  
  float f_host[10] = {1, 3, 10, 4, 2, 8, 6, 5, 9, 7};
  float *f_dev;

  float y_host[10] = {-1, -1, -1, -1, -1, 1, 1, 1, 1, 1};
  float *y_dev;
  
  float C=1.5;
  
  float alpha_host[10] = {0, 0, 0.1, 0.2, 1.5, 0, 0.2, 0.4, 1.5, 1.5 };
  float *alpha_dev;  //   l  l  l/u  l/u    u  u  l/u  l/u  l    l
    
  int expected_idx[4] = {4, 3, 8, 2};
  allocate(f_dev, 10);
  allocate(y_dev, 10);
  allocate(alpha_dev, 10);
  updateDevice(f_dev, f_host, 10);
  updateDevice(y_dev, y_host, 10); 
  updateDevice(alpha_dev, alpha_host, 10);
  
  ws->Select(f_dev, alpha_dev, y_dev, C);
  int idx[4];
  updateHost(idx, ws->idx, 4);  
  for (int i=0; i<4; i++) {
    EXPECT_EQ(idx[i], expected_idx[i]);
  }
  CUDA_CHECK(hipFree(f_dev));
  CUDA_CHECK(hipFree(y_dev));
  CUDA_CHECK(hipFree(alpha_dev));
  delete ws;
}

TEST(SmoSolverTest, KernelCacheTest) {
    int n_rows = 4;
    int n_cols = 2;
    int n_ws = n_rows;
    
    float *x_dev;
    allocate(x_dev, n_rows*n_cols);
    int *ws_idx_dev;
    allocate(ws_idx_dev, n_ws);
    
    float x_host[] = {1, 2, 1, 2, 1, 2, 3, 4};
    updateDevice(x_dev, x_host, n_rows*n_cols);
    
    int ws_idx_host[] = {0, 1, 2, 3};
    updateDevice(ws_idx_dev, ws_idx_host, n_ws);
    
    float tile_host[16];
    float tile_host_expected[] = {
      2,  4,  4,  6,
      4,  8,  8, 12,
      4,  8, 10, 14,
      6, 12, 14, 20
    };
    
    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    
    KernelCache<float> *cache = new KernelCache<float>(x_dev, n_rows, n_cols, n_ws, cublas_handle);
    float *tile_dev = cache->GetTile(ws_idx_dev);
    updateHost(tile_host, tile_dev, n_ws*n_ws);
    
    for (int i=0; i<n_ws*n_ws; i++) {
      EXPECT_EQ(tile_host[i], tile_host_expected[i]);
    }
    
    delete cache;
    n_ws = 2;
    cache = new KernelCache<float>(x_dev, n_rows, n_cols, n_ws, cublas_handle);
    ws_idx_host[1] = 3; // i.e. ws_idx_host[] = {0,3}
    updateDevice(ws_idx_dev, ws_idx_host, n_ws);
    tile_dev = cache->GetTile(ws_idx_dev);
    updateHost(tile_host, tile_dev, n_ws*n_ws);
    float tile_expected2[] = {2, 6, 6, 20};
    for (int i=0; i<n_ws*n_ws; i++) {
      EXPECT_EQ(tile_host[i], tile_expected2[i]) << i;
    }
    delete cache; 
    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
}

TEST(SmoSolverTest, SmoBlockSolveTest) {
  int n_rows = 6;
  int n_cols = 2;
  int n_ws = n_rows;
    
  float *x_dev;
  allocate(x_dev, n_rows*n_cols);
  int *ws_idx_dev;
  allocate(ws_idx_dev, n_ws);
  float *y_dev;
  allocate(y_dev, n_rows);
  float *f_dev;
  allocate(f_dev, n_rows);
  float *alpha_dev;
  allocate(alpha_dev, n_rows, true);
  float *delta_alpha_dev;
  allocate(delta_alpha_dev, n_ws, true);
  float *kernel_dev;
  allocate(kernel_dev, n_ws*n_rows);
  float *return_buff_dev;
  allocate(return_buff_dev, 2);
  
  float x_host[] = {1, 2, 1, 2, 1, 2, 1, 2, 3, 4, 4, 1};
  updateDevice(x_dev, x_host, n_rows*n_cols);
    
  int ws_idx_host[] = {0, 1, 2, 3, 4, 5};
  updateDevice(ws_idx_dev, ws_idx_host, n_ws);
  
  float y_host[] = {1, 1, 1, -1, -1, -1};
  updateDevice(y_dev, y_host, n_rows);

  float f_host[] = {-1, -1, -1, 1, 1, 1};
  updateDevice(f_dev, f_host, n_rows);

  
  float kernel_host[] = {
    2,  4,  4,  6,  5, 3,
    4,  8,  8, 12, 10, 6,
    4,  8, 10, 14, 35, 5,
    6, 12, 14, 20, 18, 8,
    5, 10, 13, 18, 17, 6,
    3,  6,  5,  8,  6, 5
  };
  
  updateDevice(kernel_dev, kernel_host, n_ws*n_rows);

  SmoBlockSolve<float, 1024><<<1, n_ws>>>(y_dev, n_ws, alpha_dev, 
      delta_alpha_dev, f_dev, kernel_dev, ws_idx_dev,
      1.5f, 1e-3f, return_buff_dev);
  
  CUDA_CHECK(hipPeekAtLastError());
  float return_buff[2];
  updateHost(return_buff, return_buff_dev, 2);
  EXPECT_LT(return_buff[0], 1e-3f) << return_buff[0];
  EXPECT_LT(return_buff[1], 100) << return_buff[1];
  
  CUDA_CHECK(hipFree(x_dev));
  CUDA_CHECK(hipFree(y_dev));
  CUDA_CHECK(hipFree(f_dev));
  CUDA_CHECK(hipFree(ws_idx_dev));
  CUDA_CHECK(hipFree(alpha_dev));
  CUDA_CHECK(hipFree(delta_alpha_dev));
  CUDA_CHECK(hipFree(kernel_dev));
  CUDA_CHECK(hipFree(return_buff_dev));
}
/*TEST_F(SmoSolverTestF, SelectWorkingSetTest) {
  ASSERT_LT(1, 2);
}*/

}; // end namespace SVM
}; // end namespace ML
