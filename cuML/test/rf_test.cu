/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "randomforest/randomforest.h"
#include <gtest/gtest.h>
#include <cuda_utils.h>
#include <test_utils.h>
#include "ml_utils.h"

namespace ML {

using namespace MLCommon;

template<typename T> // template useless for now.
struct RfInputs {
	int n_rows;
	int n_cols;
	int n_trees;
	float max_features;
	float rows_sample;
	int n_inference_rows;
	int max_depth;
	int max_leaves;
	bool bootstrap;
	int n_bins;
};

template<typename T>
::std::ostream& operator<<(::std::ostream& os, const RfInputs<T>& dims) {
	return os;
}


template<typename T>
class RfTest: public ::testing::TestWithParam<RfInputs<T> > {
protected:
	void basicTest() {

		params = ::testing::TestWithParam<RfInputs<T>>::GetParam();

		//--------------------------------------------------------
		// Random Forest - Single tree 
		//--------------------------------------------------------

		int data_len = params.n_rows * params.n_cols;
		allocate(data, data_len);
		allocate(labels, params.n_rows);

		// Populate data (assume Col major)
		std::vector<T> data_h = {30.0f, 1.0f, 2.0f, 0.0f, 10.0f, 20.0f, 10.0f, 40.0f};
		data_h.resize(data_len);
	    updateDevice(data, data_h.data(), data_len);

		// Populate labels
		labels_h = {0, 1, 0, 4};
		labels_h.resize(params.n_rows);
		ML::preprocess_labels(params.n_rows, labels_h, labels_map);
	    updateDevice(labels, labels_h.data(), params.n_rows);

		// Set selected rows: all for forest w/ single decision tree
		allocate(selected_rows, params.n_rows);
		std::vector<unsigned int> selected_rows_h = {0, 1, 2, 3};
		selected_rows_h.resize(params.n_rows);
		updateDevice(selected_rows, selected_rows_h.data(), params.n_rows);

		// Train single decision tree.
		//tree_cf = new DecisionTree::DecisionTreeClassifier();
		//std::cout << "Config: " << params.n_cols << " " << params.n_rows << " " << params.max_depth << " " << params.max_leaves  << " " << params.max_features << " " << params.bootstrap << std::endl;
		//tree_cf->fit(data, params.n_cols, params.n_rows, labels, selected_rows, params.n_rows, 
		//				params.max_depth, params.max_leaves, params.max_features, params.n_bins);
		//tree_cf->print();


 		rf_classifier = new ML::rfClassifier::rfClassifier(params.n_trees, params.bootstrap, params.max_depth, 
							params.max_leaves, 0, params.n_bins, params.rows_sample, params.max_features);
		rf_classifier->fit(data, params.n_rows, params.n_cols, labels, labels_map.size());

		// Inference data: same as train, but row major
		int inference_data_len = params.n_inference_rows * params.n_cols;
		inference_data_h = {30.0f, 10.0f, 1.0f, 20.0f, 2.0f, 10.0f, 0.0f, 40.0f};
		inference_data_h.resize(inference_data_len);


# if  0
		int single_tree_inference_data_len = params.n_cols;
		std::vector<T> single_tree_inference_data_h = {30.0f, 20.0f};
		single_tree_inference_data_h.resize(single_tree_inference_data_len);
		int predicted_val = tree_cf->predict(single_tree_inference_data_h.data());
	    std::cout << "Predicted " << predicted_val << std::endl;


		//--------------------------------------------------------
		// Random Forest
		//--------------------------------------------------------
		
 		rf_classifier = new ML::rfClassifier::rfClassifier(params.n_trees, params.bootstrap, params.max_depth , params.max_leaves, 0, params.n_bins, params.rows_sample, params.max_features);
		rf_classifier->fit(data, params.n_rows, params.n_cols, labels);

		int inference_data_len = params.n_inference_rows * params.n_cols;
		std::vector<T> inference_data_h = {30.0f, 10.0f, 1.0f, 20.0f, 2.0f, 10.0f, 0.0f, 40.0f};
		inference_data_h.resize(inference_data_len);

		predictions = rf_classifier->predict(inference_data_h.data(), params.n_inference_rows, params.n_cols, false);
		for (int i = 0; i < params.n_inference_rows; i++) {
			std::cout << "Random forest predicted " << predictions[i] << std::endl;
		}

		rf_classifier->cross_validate(inference_data_h.data(), labels_h.data(), params.n_inference_rows, params.n_cols, false);
#endif


		
    }

 	void SetUp() override {
		basicTest();
	}

	void TearDown() override {
		ML::postprocess_labels(params.n_rows, labels_h, labels_map);
		inference_data_h.clear();
		labels_h.clear();
		labels_map.clear();

		CUDA_CHECK(hipFree(labels));
		CUDA_CHECK(hipFree(data));
		CUDA_CHECK(hipFree(selected_rows));
		delete rf_classifier;
		//delete tree_cf;
	}

protected:

	RfInputs<T> params;
	T * data;
    int * labels;
	std::vector<T> inference_data_h;
	std::vector<int> labels_h;
	unsigned int * selected_rows;
	std::map<int, int> labels_map; //unique map of labels to int vals starting from 0

	
	//DecisionTree::DecisionTreeClassifier * tree_cf;
    rfClassifier * rf_classifier;

};


const std::vector<RfInputs<float> > inputsf2 = {
		  { 4, 2, 1, 1.0f, 1.0f, 4, -1, -1, false, 8},	// single tree forest, bootstrap false, unlimited depth, 8 bins
		  { 4, 2, 1, 1.0f, 1.0f, 4, 8, -1, false, 8}	// single tree forest, bootstrap false, depth of 8, 8 bins
};


//FIXME Add tests for fit and predict. Identify what would make a comparison match (similar predictions?)
// devArrMatch compares 2 device n-D arrays. See external/ml-prims/test/test_utils.h
typedef RfTest<float> RfTestF;
TEST_P(RfTestF, Fit) {
	ML::RF_metrics tmp = rf_classifier->cross_validate(inference_data_h.data(), labels_h.data(), params.n_inference_rows, params.n_cols, false);
	ASSERT_TRUE((tmp.accuracy == 1.0f));
}

INSTANTIATE_TEST_CASE_P(RfTests, RfTestF, ::testing::ValuesIn(inputsf2));


} // end namespace ML
