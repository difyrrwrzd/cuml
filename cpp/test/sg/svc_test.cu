#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuda_utils.h>
#include <gtest/gtest.h>
#include <test_utils.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include "common/cumlHandle.hpp"
#include "gram/grammatrix.h"
#include "gram/kernelmatrices.h"
#include "svm/smosolver.h"
#include "svm/svc.h"
#include "svm/workingset.h"
#include "test_utils.h"

namespace ML {
namespace SVM {
using namespace MLCommon;
using namespace GramMatrix;

template <typename math_t>
class WorkingSetTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
    allocate(f_dev, 10);
    allocate(y_dev, 10);
    allocate(alpha_dev, 10);
    updateDevice(f_dev, f_host, 10, stream);
    updateDevice(y_dev, y_host, 10, stream);
    updateDevice(alpha_dev, alpha_host, 10, stream);
  }

  void TearDown() override { CUDA_CHECK(hipStreamDestroy(stream)); }
  cumlHandle handle;
  hipStream_t stream;
  WorkingSet<math_t> *ws;

  math_t f_host[10] = {1, 3, 10, 4, 2, 8, 6, 5, 9, 7};
  math_t *f_dev;

  math_t y_host[10] = {-1, -1, -1, -1, -1, 1, 1, 1, 1, 1};
  math_t *y_dev;

  math_t C = 1.5;

  math_t alpha_host[10] = {0, 0, 0.1, 0.2, 1.5, 0, 0.2, 0.4, 1.5, 1.5};
  math_t *alpha_dev;  //   l  l  l/u  l/u    u  u  l/u  l/u  l    l

  int expected_idx[4] = {4, 3, 8, 2};
  int expected_idx2[4] = {8, 2, 4, 9};
};

typedef ::testing::Types<float, double> FloatTypes;

TYPED_TEST_CASE(WorkingSetTest, FloatTypes);

TYPED_TEST(WorkingSetTest, Init) {
  this->ws = new WorkingSet<TypeParam>(this->handle.getImpl(),
                                       this->handle.getStream(), 10);
  EXPECT_EQ(this->ws->GetSize(), 10);
  delete this->ws;

  this->ws =
    new WorkingSet<TypeParam>(this->handle.getImpl(), this->stream, 100000);
  EXPECT_EQ(this->ws->GetSize(), 1024);
  delete this->ws;
}

TYPED_TEST(WorkingSetTest, Select) {
  this->ws =
    new WorkingSet<TypeParam>(this->handle.getImpl(), this->stream, 10, 4);
  EXPECT_EQ(this->ws->GetSize(), 4);
  this->ws->SimpleSelect(this->f_dev, this->alpha_dev, this->y_dev, this->C);
  ASSERT_TRUE(devArrMatchHost(this->expected_idx, this->ws->GetIndices(),
                              this->ws->GetSize(), Compare<int>()));

  this->ws->Select(this->f_dev, this->alpha_dev, this->y_dev, this->C);
  ASSERT_TRUE(devArrMatchHost(this->expected_idx, this->ws->GetIndices(),
                              this->ws->GetSize(), Compare<int>()));
  this->ws->Select(this->f_dev, this->alpha_dev, this->y_dev, this->C);

  ASSERT_TRUE(devArrMatchHost(this->expected_idx2, this->ws->GetIndices(),
                              this->ws->GetSize(), Compare<int>()));
  delete this->ws;
}

//TYPED_TEST(WorkingSetTest, Priority) {
// See Issue #946
//}

class KernelCacheTest
  : public ::testing::TestWithParam<GramMatrix::KernelParams> {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
    cublas_handle = handle.getImpl().getCublasHandle();
    allocate(x_dev, n_rows * n_cols);
    updateDevice(x_dev, x_host, n_rows * n_cols, stream);

    allocate(ws_idx_dev, n_ws);
    updateDevice(ws_idx_dev, ws_idx_host, n_ws, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
  }

  // Naive host side kernel implementation used for comparison
  void ApplyNonlin(GramMatrix::KernelParams params) {
    switch (params.kernel) {
      case GramMatrix::LINEAR:
        break;
      case GramMatrix::POLYNOMIAL:
        for (int z = 0; z < n_rows * n_ws; z++) {
          float val = params.gamma * tile_host_expected[z] + params.coef0;
          tile_host_expected[z] = pow(val, params.degree);
        }
        break;
      case GramMatrix::TANH:
        for (int z = 0; z < n_rows * n_ws; z++) {
          float val = params.gamma * tile_host_expected[z] + params.coef0;
          tile_host_expected[z] = tanh(val);
        }
        break;
      case GramMatrix::RBF:
        for (int i = 0; i < n_ws; i++) {
          for (int j = 0; j < n_rows; j++) {
            float d = 0;
            for (int k = 0; k < n_cols; k++) {
              int idx_i = ws_idx_host[i];
              float diff = x_host[idx_i + k * n_rows] - x_host[j + k * n_rows];
              d += diff * diff;
            }
            tile_host_expected[i * n_rows + j] = exp(-params.gamma * d);
          }
        }
        break;
    }
  }
  cumlHandle handle;
  hipblasHandle_t cublas_handle;
  hipStream_t stream;

  int n_rows = 4;
  int n_cols = 2;
  int n_ws = 3;

  float *x_dev;
  int *ws_idx_dev;

  float x_host[8] = {1, 2, 3, 4, 5, 6, 7, 8};
  int ws_idx_host[4] = {0, 1, 3};
  float tile_host_expected[12] = {26, 32, 38, 44, 32, 40,
                                  48, 56, 44, 56, 68, 80};
};

TEST_P(KernelCacheTest, EvalTest) {
  GramMatrix::KernelParams params = GetParam();
  GramMatrix::GramMatrixBase<float> *kernel =
    GramMatrix::KernelFactory<float>::create(
      params, handle.getImpl().getCublasHandle());
  KernelCache<float> cache(handle.getImpl(), x_dev, n_rows, n_cols, n_ws,
                           kernel);
  float *tile_dev = cache.GetTile(ws_idx_dev);
  // apply nonlinearity on tile_host_expected
  ApplyNonlin(params);
  ASSERT_TRUE(devArrMatchHost(tile_host_expected, tile_dev, n_ws * n_ws,
                              CompareApprox<float>(1e-6f)));
  delete kernel;
}

GramMatrix::KernelParams LinearKernel{GramMatrix::LINEAR, 3, 1, 0};
GramMatrix::KernelParams PolyKernel{GramMatrix::POLYNOMIAL, 2, 1.3, 1};
GramMatrix::KernelParams TanhKernel{GramMatrix::TANH, 2, 0.5, 2.4};
GramMatrix::KernelParams RbfKernel{GramMatrix::RBF, 2, 0.5, 0};

INSTANTIATE_TEST_CASE_P(KernelTests, KernelCacheTest,
                        testing::Values(LinearKernel, PolyKernel, TanhKernel,
                                        RbfKernel));

class SmoBlockSolverTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
    cublas_handle = handle.getImpl().getCublasHandle();
    kernel = new GramMatrix::GramMatrixBase<float>(cublas_handle);
    allocate(ws_idx_dev, n_ws);
    allocate(y_dev, n_rows);
    allocate(f_dev, n_rows);
    allocate(alpha_dev, n_rows, true);
    allocate(delta_alpha_dev, n_ws, true);
    allocate(kernel_dev, n_ws * n_rows);
    allocate(return_buff_dev, 2);

    updateDevice(ws_idx_dev, ws_idx_host, n_ws, stream);
    updateDevice(y_dev, y_host, n_rows, stream);
    updateDevice(f_dev, f_host, n_rows, stream);
    updateDevice(kernel_dev, kernel_host, n_ws * n_rows, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(y_dev));
    CUDA_CHECK(hipFree(f_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
    CUDA_CHECK(hipFree(alpha_dev));
    CUDA_CHECK(hipFree(delta_alpha_dev));
    CUDA_CHECK(hipFree(kernel_dev));
    CUDA_CHECK(hipFree(return_buff_dev));
    delete kernel;
  }

  cumlHandle handle;
  hipStream_t stream;
  hipblasHandle_t cublas_handle;

  GramMatrix::GramMatrixBase<float> *kernel;
  int n_rows = 4;
  int n_cols = 2;
  int n_ws = 4;

  int *ws_idx_dev;
  float *y_dev;
  float *f_dev;
  float *alpha_dev;
  float *delta_alpha_dev;
  float *kernel_dev;
  float *return_buff_dev;

  int ws_idx_host[4] = {0, 1, 2, 3};
  float y_host[4] = {1, 1, -1, -1};
  float f_host[4] = {0.4, 0.3, 0.5, 0.1};
  float kernel_host[16] = {26, 32, 38, 44, 32, 40, 48, 56,
                           38, 48, 58, 68, 44, 56, 68, 80};
};

// test a single iteration of the block solver
TEST_F(SmoBlockSolverTest, SolveSingleTest) {
  SmoBlockSolve<float, 1024><<<1, n_ws, 0, stream>>>(
    y_dev, n_rows, alpha_dev, n_ws, delta_alpha_dev, f_dev, kernel_dev,
    ws_idx_dev, 1.5f, 1e-3f, return_buff_dev, 1);
  CUDA_CHECK(hipPeekAtLastError());

  float return_buff[2];
  updateHost(return_buff, return_buff_dev, 2, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  EXPECT_FLOAT_EQ(return_buff[0], 0.2f) << return_buff[0];
  EXPECT_EQ(return_buff[1], 1) << "Number of iterations ";

  float host_alpha[4], host_dalpha[4];
  updateHost(host_alpha, alpha_dev, n_rows, stream);
  updateHost(host_dalpha, delta_alpha_dev, n_ws, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  for (int i = 0; i < n_ws; i++) {
    EXPECT_FLOAT_EQ(y_host[i] * host_alpha[i], host_dalpha[i])
      << "alpha and delta alpha " << i;
  }
  float alpha_expected[] = {0, 0.1f, 0.1f, 0};
  for (int i = 0; i < n_rows; i++) {
    EXPECT_FLOAT_EQ(host_alpha[i], alpha_expected[i]) << "alpha " << i;
  }

  // now check if updateF works
  SmoSolver<float> smo(handle.getImpl(), 1, 0.001, kernel);

  smo.UpdateF(f_dev, n_rows, delta_alpha_dev, n_ws, kernel_dev);
  updateHost(f_host, f_dev, n_rows, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  float f_host_expected[] = {-0.2, -0.5, -0.5, -1.1};
  for (int i = 0; i < n_rows; i++) {
    EXPECT_FLOAT_EQ(f_host[i], f_host_expected[i]) << "UpdateF " << i;
  }
}

class SmoSolverTestF : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
    allocate(x_dev, n_rows * n_cols);
    allocate(ws_idx_dev, n_ws);
    allocate(y_dev, n_rows);
    allocate(y_pred, n_rows);
    allocate(f_dev, n_rows);
    allocate(alpha_dev, n_rows, true);
    allocate(delta_alpha_dev, n_ws, true);
    allocate(kernel_dev, n_ws * n_rows);
    allocate(return_buff_dev, 2);

    cublas_handle = handle.getImpl().getCublasHandle();

    updateDevice(x_dev, x_host, n_rows * n_cols, stream);
    updateDevice(ws_idx_dev, ws_idx_host, n_ws, stream);
    updateDevice(y_dev, y_host, n_rows, stream);
    updateDevice(f_dev, f_host, n_rows, stream);
    updateDevice(kernel_dev, kernel_host, n_ws * n_rows, stream);

    kernel = new GramMatrix::GramMatrixBase<float>(cublas_handle);
  }

  void TearDown() override {
    delete kernel;
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(y_dev));
    CUDA_CHECK(hipFree(y_pred));
    CUDA_CHECK(hipFree(f_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
    CUDA_CHECK(hipFree(alpha_dev));
    CUDA_CHECK(hipFree(delta_alpha_dev));
    CUDA_CHECK(hipFree(kernel_dev));
    CUDA_CHECK(hipFree(return_buff_dev));
    if (dual_coefs_d) CUDA_CHECK(hipFree(dual_coefs_d));
    if (idx_d) CUDA_CHECK(hipFree(idx_d));
    if (x_support_d) CUDA_CHECK(hipFree(x_support_d));
  }

  void checkResults(int n_coefs_exp, float *dual_coefs_exp, float b_exp,
                    float *w_exp, float *x_support_exp = nullptr,
                    int *idx_exp = nullptr, float *dual_coefs_d = nullptr,
                    float *x_support_d = nullptr, int *idx_d = nullptr,
                    float epsilon = 0.001) {
    if (dual_coefs_d == nullptr) dual_coefs_d = this->dual_coefs_d;
    if (x_support_d == nullptr) x_support_d = this->x_support_d;
    if (idx_d == nullptr) idx_d = this->idx_d;

    ASSERT_LE(n_coefs, n_coefs_exp);
    EXPECT_TRUE(devArrMatchHost(dual_coefs_exp, dual_coefs_d, n_coefs,
                                CompareApprox<float>(1e-3f)));
    float *dual_coefs_host = new float[n_coefs];
    updateHost(dual_coefs_host, dual_coefs_d, n_coefs, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    float ay = 0;
    for (int i = 0; i < n_coefs; i++) {
      ay += dual_coefs_host[i];
    }
    // Test if \sum \alpha_i y_i = 0
    EXPECT_LT(abs(ay), 1.0e-6f);

    if (x_support_exp) {
      EXPECT_TRUE(devArrMatchHost(x_support_exp, x_support_d, n_coefs * n_cols,
                                  CompareApprox<float>(1e-6f)));
    }

    if (idx_exp) {
      EXPECT_TRUE(devArrMatchHost(idx_exp, idx_d, n_coefs, Compare<int>()));
    }

    float *x_support_host = new float[n_coefs * n_cols];
    updateHost(x_support_host, x_support_d, n_coefs * n_cols, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (w_exp) {
      for (int i = 0; i < n_cols; i++) w[i] = 0;

      for (int i = 0; i < n_coefs; i++) {
        w[0] += x_support_host[i] * dual_coefs_host[i];
        w[1] += x_support_host[i + n_coefs] * dual_coefs_host[i];
      }

      for (int i = 0; i < n_cols; i++)
        EXPECT_LT(abs(w[i] - w_exp[i]), epsilon) << "@" << i;
    }

    EXPECT_LT(abs(b - b_exp), epsilon);

    delete[] dual_coefs_host;
    delete[] x_support_host;
  }

  cumlHandle handle;
  hipStream_t stream;
  GramMatrix::GramMatrixBase<float> *kernel;
  int n_rows = 6;
  const int n_cols = 2;
  int n_ws = 6;

  float *x_dev;
  int *ws_idx_dev;
  float *y_dev;
  float *y_pred;
  float *f_dev;
  float *alpha_dev;
  float *delta_alpha_dev;
  float *kernel_dev;
  float *return_buff_dev;

  float x_host[12] = {1, 2, 1, 2, 1, 2, 1, 1, 2, 2, 3, 3};
  int ws_idx_host[6] = {0, 1, 2, 3, 4, 5};
  float y_host[6] = {-1, -1, 1, -1, 1, 1};

  float f_host[6] = {1, 1, -1, 1, -1, -1};

  float kernel_host[36] = {2, 3, 3, 4, 4,  5,  3, 5, 4, 6,  5,  7,
                           3, 4, 5, 6, 7,  8,  4, 6, 6, 8,  8,  10,
                           4, 5, 7, 8, 10, 11, 5, 7, 8, 10, 11, 13};
  hipblasHandle_t cublas_handle;

  float *dual_coefs_d = nullptr;
  int n_coefs;
  int *idx_d = nullptr;
  float *x_support_d = nullptr;
  float b;
  float w[2];
};

TEST_F(SmoSolverTestF, BlockSolveTest) {
  SmoBlockSolve<float, 1024><<<1, n_ws, 0, stream>>>(
    y_dev, n_rows, alpha_dev, n_ws, delta_alpha_dev, f_dev, kernel_dev,
    ws_idx_dev, 1.0f, 1e-3f, return_buff_dev);

  CUDA_CHECK(hipPeekAtLastError());
  float return_buff[2];
  updateHost(return_buff, return_buff_dev, 2, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  EXPECT_FLOAT_EQ(return_buff[0], 2.0f) << return_buff[0];
  EXPECT_LT(return_buff[1], 100) << return_buff[1];

  // check results won't work, because it expets that GetResults was called

  float host_alpha[6], host_dalpha[6];
  updateHost(host_alpha, alpha_dev, n_rows, stream);
  updateHost(host_dalpha, delta_alpha_dev, n_ws, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  for (int i = 0; i < n_ws; i++) {
    EXPECT_FLOAT_EQ(y_host[i] * host_alpha[i], host_dalpha[i])
      << "alpha and delta alpha " << i;
  }
  float w[] = {0, 0};

  float alpha_expected[] = {0.6f, 0, 1, 1, 0, 0.6f};
  //for C=10: {0.25f, 0, 2.25f, 3.75f, 0, 1.75f};
  float ay = 0;
  for (int i = 0; i < n_rows; i++) {
    EXPECT_FLOAT_EQ(host_alpha[i], alpha_expected[i]) << "alpha " << i;
    w[0] += x_host[i] * host_alpha[i] * y_host[i];
    w[1] += x_host[i + n_rows] * host_alpha[i] * y_host[i];
    ay += host_alpha[i] * y_host[i];
  }
  EXPECT_FLOAT_EQ(ay, 0.0);
  EXPECT_FLOAT_EQ(w[0], -0.4);
  EXPECT_FLOAT_EQ(w[1], 1.2);
  // for C=10
  //EXPECT_FLOAT_EQ(w[0], -2.0);
  //EXPECT_FLOAT_EQ(w[1],  2.0);
}

TEST(SmoSolverTest, GetResultsTest) {
  cumlHandle handle;
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  handle.setStream(stream);
  auto allocator = handle.getImpl().getDeviceAllocator();

  int n_rows = 10;
  int n_cols = 2;

  device_buffer<float> x_dev(allocator, stream, n_rows * n_cols);
  float x_host[] = {1,  2,  3,  4,  5,  6,  7,  8,  9,  10,
                    11, 12, 13, 14, 15, 16, 17, 18, 19, 20};
  updateDevice(x_dev.data(), x_host, n_rows * n_cols, stream);

  float f_host[10] = {1, 3, 10, 4, 2, 8, 6, 5, 9, 7};
  device_buffer<float> f_dev(allocator, stream, n_rows);
  updateDevice(f_dev.data(), f_host, n_rows, stream);

  float y_host[10] = {-1, -1, -1, -1, -1, 1, 1, 1, 1, 1};
  device_buffer<float> y_dev(allocator, stream, n_rows);
  updateDevice(y_dev.data(), y_host, n_rows, stream);

  float C = 1.5;
  //                      l  l  l/u  l/u    u  u  l/u  l/u  l    l
  float alpha_host[10] = {0, 0, 0.1, 0.2, 1.5, 0, 0.2, 0.4, 1.5, 1.5};
  device_buffer<float> alpha_dev(allocator, stream, n_rows);
  updateDevice(alpha_dev.data(), alpha_host, n_rows, stream);

  float *dual_coefs;
  int n_coefs;
  int *idx;
  float *x_support;
  float b;

  Results<float> res(handle.getImpl(), x_dev.data(), y_dev.data(), n_rows,
                     n_cols, C);
  res.Get(alpha_dev.data(), f_dev.data(), &dual_coefs, &n_coefs, &idx,
          &x_support, &b);

  ASSERT_EQ(n_coefs, 7);

  float dual_coefs_exp[] = {-0.1, -0.2, -1.5, 0.2, 0.4, 1.5, 1.5};
  EXPECT_TRUE(devArrMatchHost(dual_coefs_exp, dual_coefs, n_coefs,
                              CompareApprox<float>(1e-6f)));

  int idx_exp[] = {2, 3, 4, 6, 7, 8, 9};
  EXPECT_TRUE(devArrMatchHost(idx_exp, idx, n_coefs, Compare<int>()));

  float x_support_exp[] = {3, 4, 5, 7, 8, 9, 10, 13, 14, 15, 17, 18, 19, 20};
  EXPECT_TRUE(devArrMatchHost(x_support_exp, x_support, n_coefs * n_cols,
                              CompareApprox<float>(1e-6f)));

  EXPECT_FLOAT_EQ(b, -6.25f);

  if (n_coefs > 0) {
    allocator->deallocate(dual_coefs, n_coefs * sizeof(float), stream);
    allocator->deallocate(idx, n_coefs * sizeof(int), stream);
    allocator->deallocate(x_support, n_coefs * n_cols * sizeof(float), stream);
  }

  // Modify the test by setting all SV's bound, then b is calculated differently
  float alpha_host2[10] = {0, 0, 1.5, 1.5, 1.5, 0, 1.5, 1.5, 1.5, 1.5};
  updateDevice(alpha_dev.data(), alpha_host2, n_rows, stream);
  res.Get(alpha_dev.data(), f_dev.data(), &dual_coefs, &n_coefs, &idx,
          &x_support, &b);
  EXPECT_FLOAT_EQ(b, -5.5f);
  CUDA_CHECK(hipStreamDestroy(stream));
}

TEST(SmoSolverTest, SmoUpdateFTest) {
  cumlHandle handle;
  hipStream_t stream;
  stream = handle.getImpl().getInternalStream(0);
  hipblasHandle_t cublas_handle = handle.getImpl().getCublasHandle();

  int n_rows = 6;
  int n_ws = 2;

  float *kernel_dev;
  allocate(kernel_dev, n_rows * n_ws);

  float *f_dev;
  allocate(f_dev, n_rows, true);

  float *delta_alpha_dev;
  allocate(delta_alpha_dev, n_ws);

  float kernel_host[] = {3, 5, 4, 6, 5, 7, 4, 5, 7, 8, 10, 11};
  updateDevice(kernel_dev, kernel_host, n_ws * n_rows, stream);

  float delta_alpha_host[] = {-0.1f, 0.1f};
  updateDevice(delta_alpha_dev, delta_alpha_host, n_ws, stream);

  GramMatrix::GramMatrixBase<float> kernel(handle.getImpl().getCublasHandle());
  SmoSolver<float> smo(handle.getImpl(), 1, 0.001, &kernel);

  smo.UpdateF(f_dev, n_rows, delta_alpha_dev, n_ws, kernel_dev);

  float f_host[6];
  updateHost(f_host, f_dev, n_rows, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  float f_host_expected[] = {0.1f, 7.4505806e-9f, 0.3f, 0.2f, 0.5f, 0.4f};
  for (int i = 0; i < n_rows; i++) {
    EXPECT_FLOAT_EQ(f_host[i], f_host_expected[i]) << "UpdateF " << i;
  }

  CUDA_CHECK(hipFree(delta_alpha_dev));
  CUDA_CHECK(hipFree(kernel_dev));
  CUDA_CHECK(hipFree(f_dev));
}

TEST_F(SmoSolverTestF, SmoSolveTest) {
  SmoSolver<float> smo(handle.getImpl(), 1, 0.001, kernel);
  smo.Solve(x_dev, n_rows, n_cols, y_dev, &dual_coefs_d, &n_coefs, &x_support_d,
            &idx_d, &b, 100, 1);

  float dual_coefs_exp[] = {-0.6, 1, -1, 0.6};
  float w_exp[] = {-0.4, 1.2};
  float x_support_exp[] = {1, 1, 2, 2, 1, 2, 2, 3};
  int idx_exp[] = {0, 2, 3, 5};
  SCOPED_TRACE("SmoSolveTest");
  checkResults(4, dual_coefs_exp, -1.8f, w_exp, x_support_exp, idx_exp);
}

TEST_F(SmoSolverTestF, SmoSolveTestLargeC) {
  float epsilon = 0.001;
  SmoSolver<float> smo(handle.getImpl(), 100, epsilon, kernel);

  smo.Solve(x_dev, n_rows, n_cols, y_dev, &dual_coefs_d, &n_coefs, &x_support_d,
            &idx_d, &b, 100, 1);

  float dual_coefs_exp[] = {-2, 4, -2, 0, 0};
  float w_exp[] = {-2, 2};

  int *idx_exp = nullptr;
  float *x_support_exp = nullptr;

  SCOPED_TRACE("SmoSolveTestLargeC");
  checkResults(4, dual_coefs_exp, -1.0f, w_exp, x_support_exp, idx_exp);
}

TEST_F(SmoSolverTestF, SmoSolvePolynomial) {
  auto nonlin =
    new GramMatrix::PolynomialKernel<float, int>(3, 1.0, 1.0, cublas_handle);
  SmoSolver<float> smo(handle.getImpl(), 1, 1e-6f, nonlin);
  smo.Solve(x_dev, n_rows, n_cols, y_dev, &dual_coefs_d, &n_coefs, &x_support_d,
            &idx_d, &b, 100, 1);

  float dual_coefs_exp[] = {-0.02556136, 0.03979708, -0.01423571};
  float *w_exp = nullptr;
  float x_support_exp[] = {1, 1, 2, 1, 2, 2};
  int idx_exp[] = {0, 2, 3};
  SCOPED_TRACE("SmoSolveTestPolynomial");
  checkResults(3, dual_coefs_exp, -1.07739149f, w_exp, x_support_exp, idx_exp);
}

TEST_F(SmoSolverTestF, SvcTest) {
  float epsilon = 0.001;
  SVC<float> svc(handle, 1.0f, epsilon, KernelParams(LINEAR));
  svc.fit(x_dev, n_rows, n_cols, y_dev);
  n_coefs = svc.n_support;
  b = svc.b;
  float dual_coefs_exp[] = {-0.6, 1, -1, 0.6};
  float w_exp[] = {-0.4, 1.2};
  float x_support_exp[] = {1, 1, 2, 2, 1, 2, 2, 3};
  int idx_exp[] = {0, 2, 3, 5};
  SCOPED_TRACE("SvcTest");
  checkResults(4, dual_coefs_exp, -1.8f, w_exp, x_support_exp, idx_exp,
               svc.dual_coefs, svc.x_support, svc.support_idx);
  // allocate a prediction buffer, then we can compare pred buffer to y_dev
  for (int i = 0; i < 3; i++) {
    svc.predict(x_dev, n_rows, n_cols, y_pred);
    EXPECT_TRUE(devArrMatch(y_dev, y_pred, n_rows, CompareApprox<float>(1e-6f)))
      << i << "th prediction";
  }
}

TEST_F(SmoSolverTestF, SvcTestPoly) {
  float epsilon = 1.0e-6;
  SVC<float> svc(handle, 1.0f, epsilon,
                 GramMatrix::KernelParams(GramMatrix::POLYNOMIAL));
  svc.fit(x_dev, n_rows, n_cols, y_dev);
  n_coefs = svc.n_support;
  b = svc.b;
  int n_coefs = 3;
  float dual_coefs_exp[] = {-0.03900895, 0.05904058, -0.02003163};
  float x_support_exp[] = {1, 1, 2, 1, 2, 2};
  int idx_exp[] = {0, 2, 3};
  SCOPED_TRACE("SvcTestPoly");
  checkResults(n_coefs, dual_coefs_exp, -0.99999959, nullptr, x_support_exp,
               idx_exp, svc.dual_coefs, svc.x_support, svc.support_idx);
}

TEST_F(SmoSolverTestF, SvcTestTanh) {
  float epsilon = 1.0e-6;
  SVC<float> svc(handle, 10.0f, epsilon,
                 GramMatrix::KernelParams(GramMatrix::TANH, 3, 0.3, 1.0));
  svc.fit(x_dev, n_rows, n_cols, y_dev);
  n_coefs = svc.n_support;
  b = svc.b;
  int n_coefs = 6;
  float dual_coefs_exp[] = {-10., -10., 10., -10., 10., 10.};
  // x_support_exp == x_host;
  int idx_exp[] = {0, 1, 2, 3, 4, 5};
  SCOPED_TRACE("SvcTestTanh");
  checkResults(n_coefs, dual_coefs_exp, -0.3927505, nullptr, x_host, idx_exp,
               svc.dual_coefs, svc.x_support, svc.support_idx);
}

TEST_F(SmoSolverTestF, SvcTestRBF) {
  float epsilon = 1.0e-6;
  SVC<float> svc(handle, 1.0f, epsilon,
                 GramMatrix::KernelParams(GramMatrix::RBF, 0, 0.15));
  svc.fit(x_dev, n_rows, n_cols, y_dev);
  n_coefs = svc.n_support;
  b = svc.b;
  int n_coefs = 6;
  float dual_coefs_exp[] = {-1., -1, 1., -1., 1, 1.};
  int idx_exp[] = {0, 1, 2, 3, 4, 5};
  SCOPED_TRACE("SvcTestRBF");
  checkResults(n_coefs, dual_coefs_exp, -0.0f, nullptr, x_host, idx_exp,
               svc.dual_coefs, svc.x_support, svc.support_idx);
}

__global__ void init_training_vectors(float *x, int n_rows, int n_cols,
                                      float *y) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n_rows * n_cols) {
    int i = tid % n_rows;
    int k = tid / n_rows;
    x[tid] = tid;
    if (k == 0) {
      y[i] = (i % 2) * 2 - 1;
    }
  }
}

TEST(SvcSolverTest, SvcTestLargeNonlin) {
  cumlHandle handle;
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  handle.setStream(stream);

  int n_rows = 1000;
  int n_cols = 780;
  float *x_dev;
  allocate(x_dev, n_rows * n_cols);
  float *y_dev;
  allocate(y_dev, n_rows);

  int TPB = 256;
  init_training_vectors<<<ceildiv(n_rows * n_cols, TPB), TPB>>>(x_dev, n_rows,
                                                                n_cols, y_dev);
  CUDA_CHECK(hipPeekAtLastError());

  float epsilon = 0.001;

  SVC<float> svc(handle, 1.0f, epsilon, KernelParams(RBF), 200, 1);
  svc.fit(x_dev, n_rows, n_cols, y_dev);

  ASSERT_LE(svc.n_support, n_rows);

  float *dual_coefs_host = new float[n_rows];
  updateHost(dual_coefs_host, svc.dual_coefs, svc.n_support, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  float ay = 0;
  for (int i = 0; i < svc.n_support; i++) {
    ay += dual_coefs_host[i];
  }
  // \sum \alpha_i y_i = 0
  EXPECT_LT(abs(ay), 1.0e-5f);

  //EXPECT_FLOAT_EQ(svc.b, -1.50995291e+09f);

  CUDA_CHECK(hipStreamDestroy(stream));
  CUDA_CHECK(hipFree(x_dev));
  CUDA_CHECK(hipFree(y_dev));
  delete[] dual_coefs_host;
}
TEST(SvcSolverTest, SvcTestLarge) {
  cumlHandle handle;
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  handle.setStream(stream);

  int n_rows = 1000;
  int n_cols = 780;
  float *x_dev;
  allocate(x_dev, n_rows * n_cols);
  float *y_dev;
  allocate(y_dev, n_rows);

  int TPB = 256;
  init_training_vectors<<<ceildiv(n_rows * n_cols, TPB), TPB>>>(x_dev, n_rows,
                                                                n_cols, y_dev);
  CUDA_CHECK(hipPeekAtLastError());

  float epsilon = 0.001;

  SVC<float> svc(handle, 1.0f, epsilon, KernelParams(), 200, 200);
  svc.fit(x_dev, n_rows, n_cols, y_dev);

  ASSERT_LE(svc.n_support, n_rows);

  float *dual_coefs_host = new float[n_rows];
  updateHost(dual_coefs_host, svc.dual_coefs, svc.n_support, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  float ay = 0;
  for (int i = 0; i < svc.n_support; i++) {
    ay += dual_coefs_host[i];
  }
  // \sum \alpha_i y_i = 0
  EXPECT_LT(abs(ay), 1.0e-5f);

  float *x_support_host = new float[n_rows * n_cols];

  updateHost(x_support_host, svc.x_support, svc.n_support * n_cols, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  float *w = new float[n_cols];
  memset(w, 0, sizeof(float) * n_cols);
  for (int i = 0; i < svc.n_support; i++) {
    for (int k = 0; k < n_cols; k++) {
      w[k] += x_support_host[i + k * svc.n_support] * dual_coefs_host[i];
    }
  }

  // for linear problems it should be unique
  for (int k = 0; k < n_cols; k++) {
    //  EXPECT_LT(abs(w[k] - 5.00001139), epsilon) << k;
  }

  //EXPECT_FLOAT_EQ(svc.b, -1.50995291e+09f);

  CUDA_CHECK(hipStreamDestroy(stream));
  CUDA_CHECK(hipFree(x_dev));
  CUDA_CHECK(hipFree(y_dev));
  delete[] dual_coefs_host;
  delete[] x_support_host;
  delete[] w;
}
};  // end namespace SVM
};  // end namespace ML
