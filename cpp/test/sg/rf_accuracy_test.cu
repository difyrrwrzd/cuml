/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuda_utils.h>
#include <gtest/gtest.h>
#include <cuml/ensemble/randomforest.hpp>
#include <random/rng.h>

namespace ML {

using namespace MLCommon;

struct RFInputs {
  int n_rows_train;
  int n_rows_test;
  uint64_t seed;
};

template <typename T>
class RFClassifierAccuracyTest : public ::testing::TestWithParam<RFInputs> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<RFInputs>::GetParam();
    rng.reset(new Random::Rng(params.seed));
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.reset(new cumlHandle(1));
    handle->setStream(stream);
    auto allocator = handle->getDeviceAllocator();
    setRFParams();
    forest = new RandomForestMetaData<T, int>;
    forest->trees = nullptr;
    X_train = (T*)allocator->allocate(params.n_rows_train * sizeof(T), stream);
    y_train = (int*)allocator->allocate(params.n_rows_train * sizeof(int),
                                        stream);
    X_test = (T*)allocator->allocate(params.n_rows_test * sizeof(T), stream);
    y_test = (int*)allocator->allocate(params.n_rows_test * sizeof(int),
                                       stream);
    y_pred = (int*)allocator->allocate(params.n_rows_test * sizeof(int),
                                       stream);
    loadData(X_train, y_train, params.n_rows_train, 1);
    loadData(X_test, y_test, params.n_rows_test, 1);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    auto allocator = handle->getDeviceAllocator();
    allocator->deallocate(X_train, params.n_rows_train * sizeof(T), stream);
    allocator->deallocate(y_train, params.n_rows_train * sizeof(int), stream);
    allocator->deallocate(X_test, params.n_rows_test * sizeof(T), stream);
    allocator->deallocate(y_test, params.n_rows_test * sizeof(int), stream);
    allocator->deallocate(y_pred, params.n_rows_test * sizeof(int), stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    handle.reset();
    rng.reset();
    delete [] forest->trees;
    delete forest;
  }

  void runTest() {
    auto& h = *(handle.get());
    fit(h, forest, X_train, params.n_rows_train, 1, y_train, 2, rfp);
    CUDA_CHECK(hipStreamSynchronize(stream));
    predict(h, forest, X_test, params.n_rows_test, 1, y_pred, false);
    auto metrics = score(h, forest, y_test, params.n_rows_test, y_pred, false);
    printf("%f ... \n", metrics.accuracy);
  }
  
 private:
  void setRFParams() {
    DecisionTree::DecisionTreeParams tree_params;
    auto algo = SPLIT_ALGO::GLOBAL_QUANTILE;
    auto sc = CRITERION::CRITERION_END;
    set_tree_params(tree_params,
                    1,     /* max_depth */
                    -1,    /* max_leaves */
                    1.0,   /* max_features */
                    16,    /* n_bins */
                    algo,  /* split_algo */
                    2,     /* min_rows_per_node */
                    0.f,   /* min_impurity_decrease */
                    false, /* bootstrap_features */
                    sc,    /* split_criterion */
                    false, /* quantile_per_tree */
                    false  /* shuffle_features */
      );
    set_all_rf_params(rfp,
                      1,    /* n_trees */
                      true, /* bootstrap */
                      1.0,  /* rows_sample */
                      -1,   /* seed */
                      1,    /* n_streams */
                      tree_params
      );
  }

  void loadData(T *X, int *y, int nrows, int ncols) {
    rng->uniform(X, nrows * ncols, T(-1.0), T(1.0), stream);
    rng->uniformInt(y, nrows, 0, 2, stream);
  }

  RFInputs params;
  RF_params rfp;
  std::shared_ptr<cumlHandle> handle;
  hipStream_t stream;
  RandomForestMetaData<T, int>* forest;
  T *X_train, *X_test;
  int *y_train, *y_test, *y_pred;
  std::shared_ptr<Random::Rng> rng;
};

const std::vector<RFInputs> inputs = {
  {800, 200, 12345ULL},
};

#define DEFINE_TEST(clz, name, testName, params)                        \
  typedef clz name;                                                     \
  TEST_P(name, Test) {                                                  \
    runTest();                                                          \
  }                                                                     \
  INSTANTIATE_TEST_CASE_P(testName, name, ::testing::ValuesIn(params))

DEFINE_TEST(RFClassifierAccuracyTest<float>, ClsTestF, RFAccuracy, inputs);
DEFINE_TEST(RFClassifierAccuracyTest<double>, ClsTestD, RFAccuracy, inputs);

#undef DEFINE_TEST

}  // end namespace ML
