/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuda_utils.h>
#include <gtest/gtest.h>
#include <test_utils.h>
#include "holtwinters/Aion.hpp"
#include "ml_utils.h"

namespace ML {

using namespace MLCommon;

#define AION_SAFE_CALL(call)                                        \
  do {                                                              \
    aion::AionStatus status = call;                                 \
    if (status != aion::AionStatus::AION_SUCCESS) {                 \
      std::cerr << "Aion error in in line " << status << std::endl; \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  } while (0)

struct HoltWintersInputs {
  int batch_size;
  int frequency;
  aion::SeasonalType seasonal;
  int start_periods;
};

template <typename T>
class HoltWintersTest : public ::testing::TestWithParam<HoltWintersInputs> {
 public:
  void basicTest() {
    params = ::testing::TestWithParam<HoltWintersInputs>::GetParam();
    int batch_size = params.batch_size;
    int frequency = params.frequency;
    aion::SeasonalType seasonal = params.seasonal;
    int start_periods = params.start_periods;

    int n = 12, h = 5;
    bool optim_alpha = true, optim_beta = true, optim_gamma = true;
    aion::ComputeMode mode = aion::ComputeMode::GPU;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    std::vector<T> dataset_h = {3.0, 2.0, 1.0, 3.0, 2.0, 1.0,
                                3.0, 2.0, 1.0, 3.0, 2.0, 1.0};

    // // initial values for alpha, beta and gamma
    // std::vector<T> alpha_h(batch_size, 0.4);
    // std::vector<T> beta_h(batch_size, 0.3);
    // std::vector<T> gamma_h(batch_size, 0.3);

    // int leveltrend_seed_len, season_seed_len, components_len;
    // int leveltrend_coef_offset, season_coef_offset;
    // int error_len;

    // AION_SAFE_CALL(aion::HoltWintersBufferSize(
    //   n, batch_size, frequency, optim_beta, optim_gamma,
    //   &leveltrend_seed_len,     // = batch_size
    //   &season_seed_len,         // = frequency*batch_size
    //   &components_len,          // = (n-w_len)*batch_size
    //   &error_len,               // = batch_size
    //   &leveltrend_coef_offset,  // = (n-wlen-1)*batch_size (last row)
    //   &season_coef_offset));  // = (n-wlen-frequency)*batch_size(last freq rows)

    allocate(dataset_d, batch_size * n);
    allocate(dataset_d_copy, batch_size * n);
    updateDevice(dataset_d_copy, dataset_h.data(), n, stream);
    // allocate(forecast_d, batch_size * h);
    // allocate(alpha_d, batch_size);
    // updateDevice(alpha_d, alpha_h.data(), batch_size, stream);
    // allocate(level_seed_d, leveltrend_seed_len);
    // allocate(level_d, components_len);

    // // if optim_beta
    // allocate(beta_d, batch_size);
    // updateDevice(beta_d, beta_h.data(), batch_size, stream);
    // allocate(trend_seed_d, leveltrend_seed_len);
    // allocate(trend_d, components_len);

    // // if optim_gamma
    // allocate(gamma_d, batch_size);
    // updateDevice(gamma_d, gamma_h.data(), batch_size, stream);
    // allocate(start_season_d, season_seed_len);
    // allocate(season_d, components_len);

    // allocate(error_d, error_len);

    // CUDA_CHECK(hipStreamSynchronize(stream));
    // CUDA_CHECK(hipStreamDestroy(stream));
    // Step 1: transpose the dataset (aion expects col major dataset)
    // MLCommon::myPrintDevVector("C", dataset_d, 12);

    CUDA_CHECK(hipStreamSynchronize(stream));
    AION_SAFE_CALL(
      aion::AionTranspose<T>(dataset_d_copy, batch_size, n, dataset_d, mode));

    // myPrintDevVector("alpha_d", alpha_d, batch_size);

    // // myPrintDevVector("Device Dataset", (const float *)dataset_d, 12);

    // // Step 2: Decompose dataset to get seed for level, trend and seasonal values
    // AION_SAFE_CALL(aion::HoltWintersDecompose<T>(
    //   dataset_d, n, batch_size, frequency, level_seed_d, trend_seed_d,
    //   start_season_d, start_periods, seasonal, mode));

    // // Step 3: Find optimal alpha, beta and gamma values (seasonal HW)
    // AION_SAFE_CALL(aion::HoltWintersOptim<T>(
    //   dataset_d, n, batch_size, frequency, level_seed_d, trend_seed_d,
    //   start_season_d, alpha_d, optim_alpha, beta_d, optim_beta, gamma_d,
    //   optim_gamma, level_d, trend_d, season_d, nullptr, error_d, nullptr,
    //   nullptr, seasonal, mode));

    // // Step 4: Do forecast
    // AION_SAFE_CALL(aion::HoltWintersForecast<T>(
    //   forecast_d, h, batch_size, frequency, level_d + leveltrend_coef_offset,
    //   trend_d + leveltrend_coef_offset, season_d + season_coef_offset, seasonal,
    //   mode));
  }

  void SetUp() override {
    AION_SAFE_CALL(aion::AionInit());
    basicTest();
  }

  void TearDown() override {
    // AION_SAFE_CALL(aion::AionDestroy());
    // CUDA_CHECK(hipFree(dataset_d));
    // CUDA_CHECK(hipFree(forecast_d));
    // CUDA_CHECK(hipFree(level_seed_d));
    // CUDA_CHECK(hipFree(trend_seed_d));
    // CUDA_CHECK(hipFree(start_season_d));
    // CUDA_CHECK(hipFree(level_d));
    // CUDA_CHECK(hipFree(trend_d));
    // CUDA_CHECK(hipFree(season_d));
    // CUDA_CHECK(hipFree(alpha_d));
    // CUDA_CHECK(hipFree(beta_d));
    // CUDA_CHECK(hipFree(gamma_d));
    // CUDA_CHECK(hipFree(error_d));
  }

 public:
  HoltWintersInputs params;
  T *dataset_d, *dataset_d_copy;
  T *forecast_d;
  T *level_seed_d, *trend_seed_d = nullptr, *start_season_d = nullptr;
  T *level_d, *trend_d = nullptr, *season_d = nullptr;
  T *alpha_d, *beta_d = nullptr, *gamma_d = nullptr;
  T *error_d;
};

const std::vector<HoltWintersInputs> inputsf = {
  {1, 3, aion::SeasonalType::ADDITIVE, 2}};

typedef HoltWintersTest<float> HoltWintersTestF;
TEST_P(HoltWintersTestF, Fit) {
  // myPrintDevVector("forecast_d", (const float *)forecast_d, 5);
  ASSERT_TRUE(true == true);
}

INSTANTIATE_TEST_CASE_P(HoltWintersTests, HoltWintersTestF,
                        ::testing::ValuesIn(inputsf));

}  // namespace ML