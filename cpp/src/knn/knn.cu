/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common/cumlHandle.hpp"

#include "knn.hpp"

#include "selection/knn.h"

#include "cuda_utils.h"
#include <hip/hip_runtime.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/Heap.h>
#include <faiss/gpu/GpuDistance.h>


#include <omp.h>
#include <vector>
#include <sstream>


namespace ML {

  void brute_force_knn(
        const cumlHandle &handle,
        float **input, int *sizes, int n_params, int D,
        float *search_items, int n,
        long *res_I, float *res_D, int k) {

    MLCommon::Selection::brute_force_knn(input, sizes, n_params, D,
        search_items, n, res_I, res_D, k, handle.getImpl().getStream());
  }


	/**
	 * Build a kNN object for training and querying a k-nearest neighbors model.
	 * @param D 	number of features in each vector
	 */
	kNN::kNN(const cumlHandle &handle, int D, bool verbose):
	        D(D), total_n(0), indices(0), verbose(verbose), owner(false) {
	    this->handle = const_cast<cumlHandle*>(&handle);
	    sizes = nullptr;
	    ptrs = nullptr;
	}

	kNN::~kNN() {

	    try {
	        if(this->owner) {
	            if(this->verbose)
	                std::cout << "Freeing kNN memory" << std::endl;
	            for(int i = 0; i < this->indices; i++) { CUDA_CHECK(hipFree(this->ptrs[i])); }
	        }

	    } catch(const std::exception &e) {
	        std::cout << "An exception occurred releasing kNN memory: " << e.what() << std::endl;
	    }

	    delete ptrs;
	    delete sizes;
	}

	void kNN::reset() {
        if(this->indices > 0) {
            this->indices = 0;
            this->total_n = 0;
        }
	}

	/**
	 * Fit a kNN model by creating separate indices for multiple given
	 * instances of kNNParams.
	 * @param input  an array of pointers to data on (possibly different) devices
	 * @param N 	 number of items in input array.
	 */
	void kNN::fit(float **input, int *sizes, int N) {

      if(this->owner)
        for(int i = 0; i < this->indices; i++) { CUDA_CHECK(hipFree(this->ptrs[i])); }

	    if(this->verbose)
	        std::cout << "N=" << N << std::endl;

	    reset();

	    // TODO: Copy pointers!
	    this->indices = N;
	    this->ptrs = (float**)malloc(N*sizeof(float*));
	    this->sizes = (int*)malloc(N*sizeof(int));

	    for(int i = 0; i < N; i++) {
	      this->ptrs[i] = input[i];
	      this->sizes[i] = sizes[i];
	    }
	}

	/**
	 * Search the kNN for the k-nearest neighbors of a set of query vectors
	 * @param search_items set of vectors to query for neighbors
	 * @param n 		   number of items in search_items
	 * @param res_I 	   pointer to device memory for returning k nearest indices
	 * @param res_D		   pointer to device memory for returning k nearest distances
	 * @param k			   number of neighbors to query
	 */
	void kNN::search(float *search_items, int n,
			long *res_I, float *res_D, int k) {

	  MLCommon::Selection::brute_force_knn(ptrs, sizes, indices, D,
	      search_items, n, res_I, res_D, k, handle->getImpl().getStream());
	}

    /**
     * Chunk a host array up into one or many GPUs (determined by the provided
     * list of gpu ids) and fit a knn model.
     *
     * @param ptr       an array in host memory to chunk over devices
     * @param n         number of elements in ptr
     * @param devices   array of device ids for chunking the ptr
     * @param n_chunks  number of elements in gpus
     * @param out       host pointer (size n) to store output
     */
    void kNN::fit_from_host(float *ptr, int n, int* devices, int n_chunks) {

        if(this->owner)
          for(int i = 0; i < this->indices; i++) { CUDA_CHECK(hipFree(this->ptrs[i])); }

        reset();

        this->owner = true;

        float **params = new float*[n_chunks];
        int *sizes = new int[n_chunks];

        MLCommon::chunk_to_device<float>(ptr, n, D, devices, params, sizes, n_chunks, handle->getImpl().getStream());

        fit(params, sizes, n_chunks);
   }
}; // end namespace


extern "C" cumlError_t knn_search(
    const cumlHandle_t handle,
    float **input, int *sizes, int n_params, int D,
    float *search_items, int n,
    long *res_I, float *res_D, int k) {

    cumlError_t status;

    ML::cumlHandle *handle_ptr;
    std::tie(handle_ptr, status) = ML::handleMap.lookupHandlePointer(handle);
    if (status == CUML_SUCCESS) {
        try {
            MLCommon::Selection::brute_force_knn(input, sizes, n_params, D,
                search_items, n,
                res_I, res_D, k,
                handle_ptr->getImpl().getStream());
        }
        catch (...) {
            status = CUML_ERROR_UNKNOWN;
        }
    }
    return status;

}

