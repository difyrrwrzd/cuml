#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <tuple>
#include <vector>

#include "batched_arima.hpp"
#include "batched_kalman.hpp"
#include "cuda_utils.h"
#include "utils.h"

#include <common/nvtx.hpp>

#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include <cuml/cuml.hpp>

#include <linalg/binary_op.h>
#include <linalg/cublas_wrappers.h>
#include <linalg/matrix_vector_op.h>
#include <metrics/batched/information_criterion.h>
#include <stats/mean.h>
#include <matrix/batched_matrix.hpp>

namespace ML {

using std::vector;

/**
 * @brief Prepare data by differencing if needed (simple and/or seasonal)
 *        and removing a trend if needed
 *
 * @note: It is assumed that d + D <= 2. This is enforced on the Python side
 *
 * @param[in]  handle      cuML handle
 * @param[out] d_out       Output. Shape (n_obs - d - D*s, batch_size) (device)
 * @param[in]  d_in        Input. Shape (n_obs, batch_size) (device)
 * @param[in]  batch_size  Number of series per batch
 * @param[in]  n_obs       Number of observations per series
 * @param[in]  d           Order of simple differences (0, 1 or 1)
 * @param[in]  D           Order of seasonal differences (0, 1 or 1)
 * @param[in]  s           Seasonal period if D > 0
 * @param[in]  intercept   Whether the model fits an intercept
 * @param[in]  d_mu        Mu array if intercept > 0
 *                         Shape (batch_size,) (device)
 */
static void _prepare_data(cumlHandle& handle, double* d_out, const double* d_in,
                          int batch_size, int n_obs, int d, int D, int s,
                          int intercept = 0, const double* d_mu = nullptr) {
  const auto stream = handle.getStream();

  // Only one difference (simple or seasonal)
  if (d + D == 1) {
    int period = d ? 1 : s;
    int tpb = (n_obs - period) > 512 ? 256 : 128;  // quick heuristics
    MLCommon::Matrix::batched_diff_kernel<<<batch_size, tpb, 0, stream>>>(
      d_in, d_out, n_obs, period);
    CUDA_CHECK(hipPeekAtLastError());
  }
  // Two differences (simple or seasonal or both)
  else if (d + D == 2) {
    int period1 = d ? 1 : s;
    int period2 = d == 2 ? 1 : s;
    int tpb = (n_obs - period1 - period2) > 512 ? 256 : 128;
    MLCommon::Matrix::
      batched_second_diff_kernel<<<batch_size, tpb, 0, stream>>>(
        d_in, d_out, n_obs, period1, period2);
    CUDA_CHECK(hipPeekAtLastError());
  }
  // If no difference and the pointers are different, copy in to out
  else if (d + D == 0 && d_in != d_out) {
    MLCommon::copy(d_out, d_in, n_obs, stream);
  }
  // Other cases: no difference and the pointers are the same, nothing to do

  // Remove trend in-place
  if (intercept) {
    MLCommon::LinAlg::matrixVectorOp(
      d_out, d_out, d_mu, batch_size, n_obs - d - D * s, false, true,
      [] __device__(double a, double b) { return a - b; }, stream);
  }
}

/**
 * @brief Helper function that will read in src0 if the given index is
 *        negative, src1 otherwise.
 * @note  This is useful when one array is the logical continuation of
 *        another and the index is expressed relatively to the second array.
 */
static __device__ double _select_read(const double* src0, int size0,
                                      const double* src1, int idx) {
  return idx < 0 ? src0[size0 + idx] : src1[idx];
}

/**
 * @brief Kernel to undifference the data with up to two levels of simple
 *        and/or seasonal differencing.
 * @note  One thread per series.
 */
template <bool double_diff>
static __global__ void _undiff_kernel(double* d_fc, const double* d_in,
                                      int num_steps, int batch_size, int n_obs,
                                      int s0, int s1 = 0) {
  int bid = blockIdx.x * blockDim.x + threadIdx.x;
  if (bid < batch_size) {
    double* b_fc = d_fc + bid * num_steps;
    const double* b_in = d_in + bid * n_obs;
    for (int i = 0; i < num_steps; i++) {
      if (!double_diff) {  // One simple or seasonal difference
        b_fc[i] += _select_read(b_in, n_obs, b_fc, i - s0);
      } else {  // Two differences (simple, seasonal or both)
        double fc_acc = _select_read(b_in, n_obs, b_fc, i - s0 - s1);
        fc_acc += _select_read(b_in, n_obs, b_fc, i - s0);
        fc_acc += _select_read(b_in, n_obs, b_fc, i - s1);
        b_fc[i] += fc_acc;
      }
    }
  }
}

/**
 * @brief Finalizes a forecast by adding the trend and/or undifferencing
 *
 * @note: It is assumed that d + D <= 2. This is enforced on the Python side
 *
 * @param[in]     handle      cuML handle
 * @param[in|out] d_fc        Forecast. Shape (num_steps, batch_size) (device)
 * @param[in]     d_in        Original data. Shape (n_obs, batch_size) (device)
 * @param[in]     num_steps   Number of steps forecasted
 * @param[in]     batch_size  Number of series per batch
 * @param[in]     n_obs       Number of observations per series
 * @param[in]     d           Order of simple differences (0, 1 or 1)
 * @param[in]     D           Order of seasonal differences (0, 1 or 1)
 * @param[in]     s           Seasonal period if D > 0
 * @param[in]     intercept   Whether the model fits an intercept
 * @param[in]     d_mu        Mu array if intercept > 0
 *                            Shape (batch_size,) (device)
 */
static void _finalize_forecast(cumlHandle& handle, double* d_fc,
                               const double* d_in, int num_steps,
                               int batch_size, int n_obs, int d, int D, int s,
                               int intercept = 0,
                               const double* d_mu = nullptr) {
  const auto stream = handle.getStream();

  // Add the trend in-place
  if (intercept) {
    MLCommon::LinAlg::matrixVectorOp(
      d_fc, d_fc, d_mu, batch_size, num_steps, false, true,
      [] __device__(double a, double b) { return a + b; }, stream);
  }

  // Undifference
  constexpr int TPB = 64;  // One thread per series -> avoid big blocks
  if (d + D == 1) {
    _undiff_kernel<false>
      <<<MLCommon::ceildiv<int>(batch_size, TPB), TPB, 0, stream>>>(
        d_fc, d_in, num_steps, batch_size, n_obs, d ? 1 : s);
    CUDA_CHECK(hipPeekAtLastError());
  } else if (d + D == 2) {
    _undiff_kernel<true>
      <<<MLCommon::ceildiv<int>(batch_size, TPB), TPB, 0, stream>>>(
        d_fc, d_in, num_steps, batch_size, n_obs, d ? 1 : s, d == 2 ? 1 : s);
    CUDA_CHECK(hipPeekAtLastError());
  }
}

void residual(cumlHandle& handle, const double* d_y, int batch_size, int n_obs,
              int p, int d, int q, int P, int D, int Q, int s, int intercept,
              double* d_params, double* d_vs, bool trans) {
  ML::PUSH_RANGE(__func__);
  std::vector<double> loglike = std::vector<double>(batch_size);
  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_params, loglike.data(), d_vs, trans);
  ML::POP_RANGE();
}

void forecast(cumlHandle& handle, int num_steps, int p, int d, int q, int P,
              int D, int Q, int s, int intercept, int batch_size, int n_obs,
              const double* d_y, const double* d_y_prep, double* d_vs,
              double* d_params, double* d_y_fc) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  const auto stream = handle.getStream();

  // Unpack parameters
  double *d_mu, *d_ar, *d_ma, *d_sar, *d_sma;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_ar, &d_ma,
                  &d_sar, &d_sma, intercept, &d_mu);
  unpack(d_params, d_mu, d_ar, d_ma, d_sar, d_sma, batch_size, p, q, P, Q,
         intercept, stream);

  int ld_yprep = n_obs - d - D * s;

  // Prepare data if given unprepared data
  double* yprep = nullptr;
  if (d_y_prep == nullptr) {
    if (intercept + d + D == 0)
      d_y_prep = d_y;
    else {
      yprep = (double*)allocator->allocate(
        ld_yprep * batch_size * sizeof(double), stream);
      _prepare_data(handle, yprep, d_y, batch_size, n_obs, d, 0, 0, intercept,
                    d_mu);
      d_y_prep = yprep;
    }
  }

  const auto counting = thrust::make_counting_iterator(0);

  // Copy data into temporary work arrays
  double* d_y_ =
    (double*)allocator->allocate((p + num_steps) * batch_size, stream);
  double* d_vs_ =
    (double*)allocator->allocate((q + num_steps) * batch_size, stream);
  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     if (p > 0) {
                       for (int ip = 0; ip < p; ip++) {
                         d_y_[(p + num_steps) * bid + ip] =
                           d_y_prep[ld_yprep * bid + ld_yprep - p + ip];
                       }
                     }
                     if (q > 0) {
                       for (int iq = 0; iq < q; iq++) {
                         d_vs_[(q + num_steps) * bid + iq] =
                           d_vs[ld_yprep * bid + ld_yprep - q + iq];
                       }
                     }
                   });

  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     for (int i = 0; i < num_steps; i++) {
                       auto it = num_steps * bid + i;
                       d_y_fc[it] = 0.0;
                       if (p > 0) {
                         double dot_ar_y = 0.0;
                         for (int ip = 0; ip < p; ip++) {
                           dot_ar_y += d_ar[p * bid + ip] *
                                       d_y_[(p + num_steps) * bid + i + ip];
                         }
                         d_y_fc[it] += dot_ar_y;
                       }
                       if (q > 0 && i < q) {
                         double dot_ma_y = 0.0;
                         for (int iq = 0; iq < q; iq++) {
                           dot_ma_y += d_ma[q * bid + iq] *
                                       d_vs_[(q + num_steps) * bid + i + iq];
                         }
                         d_y_fc[it] += dot_ma_y;
                       }
                       if (p > 0) {
                         d_y_[(p + num_steps) * bid + i + p] = d_y_fc[it];
                       }
                     }
                   });

  _finalize_forecast(handle, d_y_fc, d_y, num_steps, batch_size, n_obs, d, D, s,
                     intercept, d_mu);

  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_ar, d_ma,
                    d_sar, d_sma, intercept, d_mu);
  allocator->deallocate(d_y_, (p + num_steps) * batch_size, stream);
  allocator->deallocate(d_vs_, (q + num_steps) * batch_size, stream);
  if (yprep != nullptr)
    allocator->deallocate(yprep, ld_yprep * batch_size * sizeof(double),
                          stream);
  ML::POP_RANGE();
}

void predict_in_sample(cumlHandle& handle, const double* d_y, int batch_size,
                       int n_obs, int p, int d, int q, int P, int D, int Q,
                       int s, int intercept, double* d_params, double* d_vs,
                       double* d_y_p) {
  ML::PUSH_RANGE(__func__);
  residual(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s, intercept,
           d_params, d_vs, false);
  auto stream = handle.getStream();
  double* d_y_diff;

  ///TODO: update for seasonality
  if (d == 0) {
    auto counting = thrust::make_counting_iterator(0);
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       for (int i = 0; i < n_obs; i++) {
                         int it = bid * n_obs + i;
                         d_y_p[it] = d_y[it] - d_vs[it];
                       }
                     });
  } else {
    ///TODO: compute diff with _prepare_data
    d_y_diff = (double*)handle.getDeviceAllocator()->allocate(
      sizeof(double) * batch_size * (n_obs - 1), handle.getStream());
    auto counting = thrust::make_counting_iterator(0);
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       for (int i = 0; i < n_obs - 1; i++) {
                         int it = bid * n_obs + i;
                         int itd = bid * (n_obs - 1) + i;
                         // note: d_y[it] + (d_y[it + 1] - d_y[it]) - d_vs[itd]
                         //    -> d_y[it+1] - d_vs[itd]
                         d_y_p[it] = d_y[it + 1] - d_vs[itd];
                         d_y_diff[itd] = d_y[it + 1] - d_y[it];
                         if (intercept)
                           d_y_diff[itd] -= d_params[(p + q + intercept) * bid];
                       }
                     });
  }

  // due to `differencing` we need to forecast a single step to make the
  // in-sample prediction the same length as the original signal.
  if (d == 1) {
    double* d_y_fc = (double*)handle.getDeviceAllocator()->allocate(
      sizeof(double) * batch_size, handle.getStream());
    forecast(handle, 1, p, d, q, P, D, Q, s, intercept, batch_size, n_obs, d_y,
             d_y_diff, d_vs, d_params, d_y_fc);

    // append forecast to end of in-sample prediction
    auto counting = thrust::make_counting_iterator(0);
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_y_p[bid * n_obs + (n_obs - 1)] = d_y_fc[bid];
                     });
    handle.getDeviceAllocator()->deallocate(
      d_y_diff, sizeof(double) * batch_size * (n_obs - 1), handle.getStream());
    handle.getDeviceAllocator()->deallocate(d_y_fc, sizeof(double) * batch_size,
                                            handle.getStream());
  }
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, int p, int d, int q, int P, int D, int Q, int s,
                     int intercept, double* d_mu, double* d_ar, double* d_ma,
                     double* d_sar, double* d_sma, double* loglike,
                     double* d_vs, bool trans, bool host_loglike) {
  using std::get;

  ML::PUSH_RANGE(__func__);

  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double *d_Tar, *d_Tma, *d_Tsar, *d_Tsma;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_Tar, &d_Tma,
                  &d_Tsar, &d_Tsma);

  if (trans) {
    batched_jones_transform(handle, p, q, P, Q, batch_size, false, d_ar, d_ma,
                            d_sar, d_sma, d_Tar, d_Tma, d_Tsar, d_Tsma);
  } else {
    // non-transformed case: just use original parameters
    CUDA_CHECK(hipMemcpyAsync(d_Tar, d_ar, sizeof(double) * batch_size * p,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tma, d_ma, sizeof(double) * batch_size * q,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tsar, d_sar, sizeof(double) * batch_size * P,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tsma, d_sma, sizeof(double) * batch_size * Q,
                               hipMemcpyDeviceToDevice, stream));
  }

  if (d + D + intercept == 0) {
    batched_kalman_filter(handle, d_y, n_obs, d_Tar, d_Tma, d_Tsar, d_Tsma, p,
                          q, P, Q, batch_size, loglike, d_vs);
  } else {
    double* d_y_prep = (double*)allocator->allocate(
      batch_size * (n_obs - d - s * D) * sizeof(double), stream);

    _prepare_data(handle, d_y_prep, d_y, batch_size, n_obs, d, D, s, intercept,
                  d_mu);

    batched_kalman_filter(handle, d_y_prep, n_obs - d, d_Tar, d_Tma, d_Tsar,
                          d_Tsma, p, q, P, Q, batch_size, loglike, d_vs);

    allocator->deallocate(
      d_y_prep, sizeof(double) * batch_size * (n_obs - d - s * D), stream);
  }
  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_Tar, d_Tma,
                    d_Tsar, d_Tsma);
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, int p, int d, int q, int P, int D, int Q, int s,
                     int intercept, double* d_params, double* loglike,
                     double* d_vs, bool trans, bool host_loglike) {
  ML::PUSH_RANGE(__func__);

  // unpack parameters
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double *d_mu, *d_ar, *d_ma, *d_sar, *d_sma;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_ar, &d_ma,
                  &d_sar, &d_sma, intercept, &d_mu);
  unpack(d_params, d_mu, d_ar, d_ma, d_sar, d_sma, batch_size, p, q, P, Q,
         intercept, stream);

  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_mu, d_ar, d_ma, d_sar, d_sma, loglike, d_vs,
                  trans, host_loglike);

  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_ar, d_ma,
                    d_sar, d_sma, intercept, d_mu);
  ML::POP_RANGE();
}

void information_criterion(cumlHandle& handle, const double* d_y,
                           int batch_size, int n_obs, int p, int d, int q,
                           int P, int D, int Q, int s, int intercept,
                           double* d_mu, double* d_ar, double* d_ma,
                           double* d_sar, double* d_sma, double* ic,
                           int ic_type) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double* d_vs = (double*)allocator->allocate(
    sizeof(double) * (n_obs - d - s * D) * batch_size, stream);
  double* d_ic =
    (double*)allocator->allocate(sizeof(double) * batch_size, stream);

  /* Compute log-likelihood in d_ic */
  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_mu, d_ar, d_ma, d_sar, d_sma, d_ic, d_vs, true,
                  false);

  /* Compute information criterion from log-likelihood and base term */
  MLCommon::Metrics::Batched::information_criterion(
    d_ic, d_ic, static_cast<MLCommon::Metrics::IC_Type>(ic_type),
    p + q + P + Q + intercept, batch_size, n_obs, stream);

  /* Transfer information criterion device -> host */
  MLCommon::updateHost(ic, d_ic, batch_size, stream);

  allocator->deallocate(d_vs, sizeof(double) * (n_obs - d - s * D) * batch_size,
                        stream);
  allocator->deallocate(d_ic, sizeof(double) * batch_size, stream);
  ML::POP_RANGE();
}

/**
 * Auxiliary function of _start_params: least square approximation of an
 * ARMA model (with or without seasonality)
 */
static void _arma_least_squares(
  cumlHandle& handle, double* d_ar, double* d_ma,
  const MLCommon::Matrix::BatchedMatrix<double>& bm_y, int p, int q,
  int s = 1) {
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  // Initialize params
  if (p)
    CUDA_CHECK(
      hipMemsetAsync(d_ar, 0, sizeof(double) * p * batch_size, stream));
  if (q)
    CUDA_CHECK(
      hipMemsetAsync(d_ma, 0, sizeof(double) * q * batch_size, stream));

  int ps = p * s, qs = q * s;
  int p_ar = 2 * qs;
  int r = std::max(p_ar + qs, ps);

  if ((q && p_ar >= n_obs - p_ar) || p + q >= n_obs - r) {
    // Too few observations for the estimate, keep 0
    return;
  }

  /* Matrix formed by lag matrices of y and the residuals respectively,
   * side by side. The left side will be used to estimate AR, the right
   * side to estimate MA */
  MLCommon::Matrix::BatchedMatrix<double> bm_ls_ar_res(
    n_obs - r, p + q, batch_size, cublas_handle, allocator, stream, false);
  ///TODO: double-check these
  int ar_offset = r - ps;
  int res_offset = (ps < p_ar + qs) ? 0 : ps - p_ar - qs;

  // Get residuals from an AR(p_ar) model to estimate the MA parameters
  if (q) {
    // Create lagged y
    int ls_height = n_obs - p_ar;
    MLCommon::Matrix::BatchedMatrix<double> bm_ls =
      MLCommon::Matrix::b_lagged_mat(bm_y, p_ar);

    /* Matrix for the initial AR fit, initialized by copy of y
     * (note: this is because gels works in-place ; the matrix has larger
     *  dimensions than the actual AR fit) */
    MLCommon::Matrix::BatchedMatrix<double> bm_ar_fit =
      MLCommon::Matrix::b_2dcopy(bm_y, p_ar, 0, ls_height, 1);

    // Residual, initialized as offset y to avoid one kernel call
    MLCommon::Matrix::BatchedMatrix<double> bm_residual(
      ls_height, 1, batch_size, cublas_handle, allocator, stream, false);
    MLCommon::copy(bm_residual.raw_data(), bm_ar_fit.raw_data(),
                   ls_height * batch_size, stream);

    // Initial AR fit
    MLCommon::Matrix::b_gels(bm_ls, bm_ar_fit);

    // Compute residual (technically a gemv)
    MLCommon::Matrix::b_gemm(false, false, ls_height, 1, p_ar, -1.0, bm_ls,
                             bm_ar_fit, 1.0, bm_residual);

    // Lags of the residual
    MLCommon::Matrix::b_lagged_mat(bm_residual, bm_ls_ar_res, q, n_obs - r,
                                   res_offset, (n_obs - r) * p, s);
  }

  // Lags of y
  MLCommon::Matrix::b_lagged_mat(bm_y, bm_ls_ar_res, p, n_obs - r, ar_offset,
                                 0, s);

  /* Initializing the vector for the ARMA fit
   * (note: also in-place as described for AR fit) */
  MLCommon::Matrix::BatchedMatrix<double> bm_arma_fit =
    MLCommon::Matrix::b_2dcopy(bm_y, r, 0, n_obs - r, 1);

  // ARMA fit
  MLCommon::Matrix::b_gels(bm_ls_ar_res, bm_arma_fit);

  /* Copy the results in the AR and MA parameters batched vectors
   * Note: calling directly the kernel as there is not yet a way to wrap
   *       existing device pointers in a batched matrix */
  if (p) {
    MLCommon::Matrix::batched_2dcopy_kernel<<<batch_size, p, 0, stream>>>(
      bm_arma_fit.raw_data(), d_ar, 0, 0, n_obs - r, 1, p, 1);
    CUDA_CHECK(hipPeekAtLastError());
  }
  if (q) {
    MLCommon::Matrix::batched_2dcopy_kernel<<<batch_size, q, 0, stream>>>(
      bm_arma_fit.raw_data(), d_ma, p, 0, n_obs - r, 1, q, 1);
    CUDA_CHECK(hipPeekAtLastError());
  }
}

/**
 * Auxiliary function of estimate_x0: compute the starting parameters for
 * the series pre-processed by estimate_x0
 *
 * @note: bm_y can be mutated! estimate_x0 has already created a copy.
 */
static void _start_params(cumlHandle& handle, double* d_mu, double* d_ar,
                          double* d_ma, double* d_sar, double* d_sma,
                          MLCommon::Matrix::BatchedMatrix<double>& bm_y, int p,
                          int q, int P, int Q, int s, int intercept) {
  auto stream = handle.getStream();

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  if (intercept) {
    // Compute means and write them in mu
    MLCommon::Stats::mean(d_mu, bm_y.raw_data(), batch_size, n_obs, false,
                          false, stream);

    // Center the series around their means in-place
    MLCommon::LinAlg::matrixVectorOp(
      bm_y.raw_data(), bm_y.raw_data(), d_mu, batch_size, n_obs, false, true,
      [] __device__(double a, double b) { return a - b; }, stream);
  }

  // Estimate an ARMA fit without seasonality
  if (p + q) _arma_least_squares(handle, d_ar, d_ma, bm_y, p, q);

  // Estimate a seasonal ARMA fit independantly
  if (P + Q) _arma_least_squares(handle, d_sar, d_sma, bm_y, P, Q, s);
}

void estimate_x0(cumlHandle& handle, double* d_mu, double* d_ar, double* d_ma,
                 double* d_sar, double* d_sma, const double* d_y,
                 int batch_size, int n_obs, int p, int d, int q, int P, int D,
                 int Q, int s, int intercept) {
  ML::PUSH_RANGE(__func__);
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();

  // Difference if necessary, copy otherwise
  MLCommon::Matrix::BatchedMatrix<double> bm_yd(
    n_obs - d - s * D, 1, batch_size, cublas_handle, allocator, stream, false);
  _prepare_data(handle, bm_yd.raw_data(), d_y, batch_size, n_obs, d, D, s);
  // Note: mu is not known yet! We just want to difference the data

  // Do the computation of the initial parameters
  _start_params(handle, d_mu, d_ar, d_ma, d_sar, d_sma, bm_yd, p, q, P, Q, s,
                intercept);
  ML::POP_RANGE();
}

}  // namespace ML
