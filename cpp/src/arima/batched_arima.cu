#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include <common/nvtx.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include "arima_helpers.cuh"
#include "batched_arima.hpp"
#include "batched_kalman.hpp"

#include <cuml/cuml.hpp>

#include <linalg/binary_op.h>
#include <linalg/cublas_wrappers.h>
#include <linalg/matrix_vector_op.h>
#include <metrics/batched/information_criterion.h>
#include <stats/mean.h>
#include <matrix/batched_matrix.hpp>
#include "cuda_utils.h"
#include "utils.h"

namespace ML {

/**
 * @brief Prepare data by differencing if needed (simple and/or seasonal)
 *        and removing a trend if needed
 *
 * @note: It is assumed that d + D <= 2. This is enforced on the Python side
 *
 * @param[in]  handle      cuML handle
 * @param[out] d_out       Output. Shape (n_obs - d - D*s, batch_size) (device)
 * @param[in]  d_in        Input. Shape (n_obs, batch_size) (device)
 * @param[in]  batch_size  Number of series per batch
 * @param[in]  n_obs       Number of observations per series
 * @param[in]  d           Order of simple differences (0, 1 or 1)
 * @param[in]  D           Order of seasonal differences (0, 1 or 1)
 * @param[in]  s           Seasonal period if D > 0
 * @param[in]  intercept   Whether the model fits an intercept
 * @param[in]  d_mu        Mu array if intercept > 0
 *                         Shape (batch_size,) (device)
 */
static void _prepare_data(cumlHandle& handle, double* d_out, const double* d_in,
                          int batch_size, int n_obs, int d, int D, int s,
                          int intercept = 0, const double* d_mu = nullptr) {
  const auto stream = handle.getStream();

  // Only one difference (simple or seasonal)
  if (d + D == 1) {
    int period = d ? 1 : s;
    int tpb = (n_obs - period) > 512 ? 256 : 128;  // quick heuristics
    MLCommon::Matrix::batched_diff_kernel<<<batch_size, tpb, 0, stream>>>(
      d_in, d_out, n_obs, period);
    CUDA_CHECK(hipPeekAtLastError());
  }
  // Two differences (simple or seasonal or both)
  else if (d + D == 2) {
    int period1 = d ? 1 : s;
    int period2 = d == 2 ? 1 : s;
    int tpb = (n_obs - period1 - period2) > 512 ? 256 : 128;
    MLCommon::Matrix::
      batched_second_diff_kernel<<<batch_size, tpb, 0, stream>>>(
        d_in, d_out, n_obs, period1, period2);
    CUDA_CHECK(hipPeekAtLastError());
  }
  // If no difference and the pointers are different, copy in to out
  else if (d + D == 0 && d_in != d_out) {
    MLCommon::copy(d_out, d_in, n_obs * batch_size, stream);
  }
  // Other cases: no difference and the pointers are the same, nothing to do

  // Remove trend in-place
  if (intercept) {
    MLCommon::LinAlg::matrixVectorOp(
      d_out, d_out, d_mu, batch_size, n_obs - d - D * s, false, true,
      [] __device__(double a, double b) { return a - b; }, stream);
  }
}

/**
 * @brief Helper function that will read in src0 if the given index is
 *        negative, src1 otherwise.
 * @note  This is useful when one array is the logical continuation of
 *        another and the index is expressed relatively to the second array.
 */
static __device__ double _select_read(const double* src0, int size0,
                                      const double* src1, int idx) {
  return idx < 0 ? src0[size0 + idx] : src1[idx];
}

/**
 * @brief Kernel to undifference the data with up to two levels of simple
 *        and/or seasonal differencing.
 * @note  One thread per series.
 */
template <bool double_diff>
static __global__ void _undiff_kernel(double* d_fc, const double* d_in,
                                      int num_steps, int batch_size, int in_ld,
                                      int n_in, int s0, int s1 = 0) {
  int bid = blockIdx.x * blockDim.x + threadIdx.x;
  if (bid < batch_size) {
    double* b_fc = d_fc + bid * num_steps;
    const double* b_in = d_in + bid * in_ld;
    for (int i = 0; i < num_steps; i++) {
      if (!double_diff) {  // One simple or seasonal difference
        b_fc[i] += _select_read(b_in, n_in, b_fc, i - s0);
      } else {  // Two differences (simple, seasonal or both)
        double fc_acc = -_select_read(b_in, n_in, b_fc, i - s0 - s1);
        fc_acc += _select_read(b_in, n_in, b_fc, i - s0);
        fc_acc += _select_read(b_in, n_in, b_fc, i - s1);
        b_fc[i] += fc_acc;
      }
    }
  }
}

/**
 * @brief Finalizes a forecast by adding the trend and/or undifferencing
 *
 * @note: It is assumed that d + D <= 2. This is enforced on the Python side
 *
 * @param[in]     handle      cuML handle
 * @param[in|out] d_fc        Forecast. Shape (num_steps, batch_size) (device)
 * @param[in]     d_in        Original data. Shape (n_obs, batch_size) (device)
 * @param[in]     num_steps   Number of steps forecasted
 * @param[in]     batch_size  Number of series per batch
 * @param[in]     in_ld       Leading dimension of d_in
 * @param[in]     n_in        Number of observations/predictions in d_in
 * @param[in]     d           Order of simple differences (0, 1 or 1)
 * @param[in]     D           Order of seasonal differences (0, 1 or 1)
 * @param[in]     s           Seasonal period if D > 0
 * @param[in]     intercept   Whether the model fits an intercept
 * @param[in]     d_mu        Mu array if intercept > 0
 *                            Shape (batch_size,) (device)
 */
static void _finalize_forecast(cumlHandle& handle, double* d_fc,
                               const double* d_in, int num_steps,
                               int batch_size, int in_ld, int n_in, int d,
                               int D, int s, int intercept = 0,
                               const double* d_mu = nullptr) {
  const auto stream = handle.getStream();

  // Add the trend in-place
  if (intercept) {
    MLCommon::LinAlg::matrixVectorOp(
      d_fc, d_fc, d_mu, batch_size, num_steps, false, true,
      [] __device__(double a, double b) { return a + b; }, stream);
  }

  // Undifference
  constexpr int TPB = 64;  // One thread per series -> avoid big blocks
  if (d + D == 1) {
    _undiff_kernel<false>
      <<<MLCommon::ceildiv<int>(batch_size, TPB), TPB, 0, stream>>>(
        d_fc, d_in, num_steps, batch_size, in_ld, n_in, d ? 1 : s);
    CUDA_CHECK(hipPeekAtLastError());
  } else if (d + D == 2) {
    _undiff_kernel<true>
      <<<MLCommon::ceildiv<int>(batch_size, TPB), TPB, 0, stream>>>(
        d_fc, d_in, num_steps, batch_size, in_ld, n_in, d ? 1 : s,
        d == 2 ? 1 : s);
    CUDA_CHECK(hipPeekAtLastError());
  }
}

/// TODO: remove one of the two residual() versions

void residual(cumlHandle& handle, const double* d_y, int batch_size, int n_obs,
              int p, int d, int q, int P, int D, int Q, int s, int intercept,
              const double* d_params, double* d_vs, bool trans, int fc_steps,
              double* d_fc) {
  ML::PUSH_RANGE(__func__);
  std::vector<double> loglike = std::vector<double>(batch_size);
  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_params, loglike.data(), d_vs, trans, true,
                  fc_steps, d_fc);
  ML::POP_RANGE();
}

void residual(cumlHandle& handle, const double* d_y, int batch_size, int n_obs,
              int p, int d, int q, int P, int D, int Q, int s, int intercept,
              const double* d_mu, const double* d_ar, const double* d_ma,
              const double* d_sar, const double* d_sma, double* d_vs,
              bool trans, int fc_steps, double* d_fc) {
  ML::PUSH_RANGE(__func__);
  std::vector<double> loglike = std::vector<double>(batch_size);
  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_mu, d_ar, d_ma, d_sar, d_sma, loglike.data(),
                  d_vs, trans, true, fc_steps, d_fc);
  ML::POP_RANGE();
}

/// TODO: see if simplifications can be made because of the recent changes
void predict(cumlHandle& handle, const double* d_y, int batch_size, int n_obs,
             int start, int end, int p, int d, int q, int P, int D, int Q,
             int s, int intercept, const double* d_params, double* d_vs,
             double* d_y_p) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  const auto stream = handle.getStream();

  // Unpack parameters
  double *d_mu, *d_ar, *d_ma, *d_sar, *d_sma;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_ar, &d_ma,
                  &d_sar, &d_sma, intercept, &d_mu);
  unpack(d_params, d_mu, d_ar, d_ma, d_sar, d_sma, batch_size, p, q, P, Q,
         intercept, stream);

  // Prepare data
  int d_sD = d + D * s;
  int ld_yprep = n_obs - d_sD;
  double* d_y_prep = (double*)allocator->allocate(
    ld_yprep * batch_size * sizeof(double), stream);
  _prepare_data(handle, d_y_prep, d_y, batch_size, n_obs, d, D, s, intercept,
                d_mu);

  // Create temporary array for the forecasts
  int num_steps = std::max(end - n_obs, 0);
  double* d_y_fc = nullptr;
  if (num_steps) {
    d_y_fc = (double*)allocator->allocate(
      num_steps * batch_size * sizeof(double), stream);
  }

  // Compute the residual and forecast - provide already prepared data and
  // extracted parameters
  residual(handle, d_y_prep, batch_size, n_obs - d_sD, p, 0, q, P, 0, Q, s, 0,
           nullptr, d_ar, d_ma, d_sar, d_sma, d_vs, false, num_steps, d_y_fc);

  auto counting = thrust::make_counting_iterator(0);
  int predict_ld = end - start;

  //
  // In-sample prediction
  //

  int p_start = std::max(start, d_sD);
  int p_end = std::min(n_obs, end);

  // The prediction loop starts by filling undefined predictions with NaN,
  // then computes the predictions from the observations and residuals
  if (start < n_obs) {
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_y_p[0] = 0.0;
                       for (int i = 0; i < d_sD - start; i++) {
                         d_y_p[bid * predict_ld + i] = nan("");
                       }
                       for (int i = p_start; i < p_end; i++) {
                         d_y_p[bid * predict_ld + i - start] =
                           d_y[bid * n_obs + i] -
                           d_vs[bid * ld_yprep + i - d_sD];
                       }
                     });
  }

  //
  // Finalize out-of-sample forecast and copy in-sample predictions
  //

  if (num_steps) {
    // Add trend and/or undiff
    _finalize_forecast(handle, d_y_fc, d_y, num_steps, batch_size, n_obs, n_obs,
                       d, D, s, intercept, d_mu);

    // Copy forecast in d_y_p
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       for (int i = 0; i < num_steps; i++) {
                         d_y_p[bid * predict_ld + n_obs - start + i] =
                           d_y_fc[num_steps * bid + i];
                       }
                     });

    allocator->deallocate(d_y_fc, num_steps * batch_size * sizeof(double),
                          stream);
  }

  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_ar, d_ma,
                    d_sar, d_sma, intercept, d_mu);
  allocator->deallocate(d_y_prep, ld_yprep * batch_size * sizeof(double),
                        stream);
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, int p, int d, int q, int P, int D, int Q, int s,
                     int intercept, const double* d_mu, const double* d_ar,
                     const double* d_ma, const double* d_sar,
                     const double* d_sma, double* loglike, double* d_vs,
                     bool trans, bool host_loglike, int fc_steps,
                     double* d_fc) {
  ML::PUSH_RANGE(__func__);

  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double *d_Tar, *d_Tma, *d_Tsar, *d_Tsma;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_Tar, &d_Tma,
                  &d_Tsar, &d_Tsma);

  if (trans) {
    batched_jones_transform(handle, p, q, P, Q, batch_size, false, d_ar, d_ma,
                            d_sar, d_sma, d_Tar, d_Tma, d_Tsar, d_Tsma);
  } else {
    // non-transformed case: just use original parameters
    CUDA_CHECK(hipMemcpyAsync(d_Tar, d_ar, sizeof(double) * batch_size * p,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tma, d_ma, sizeof(double) * batch_size * q,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tsar, d_sar, sizeof(double) * batch_size * P,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tsma, d_sma, sizeof(double) * batch_size * Q,
                               hipMemcpyDeviceToDevice, stream));
  }

  if (d + D + intercept == 0) {
    batched_kalman_filter(handle, d_y, n_obs, d_Tar, d_Tma, d_Tsar, d_Tsma, p,
                          q, P, Q, s, batch_size, loglike, d_vs, host_loglike,
                          false, fc_steps, d_fc);
  } else {
    double* d_y_prep = (double*)allocator->allocate(
      batch_size * (n_obs - d - s * D) * sizeof(double), stream);

    _prepare_data(handle, d_y_prep, d_y, batch_size, n_obs, d, D, s, intercept,
                  d_mu);

    batched_kalman_filter(handle, d_y_prep, n_obs - d - s * D, d_Tar, d_Tma,
                          d_Tsar, d_Tsma, p, q, P, Q, s, batch_size, loglike,
                          d_vs, host_loglike, false, fc_steps, d_fc);

    allocator->deallocate(
      d_y_prep, sizeof(double) * batch_size * (n_obs - d - s * D), stream);
  }
  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_Tar, d_Tma,
                    d_Tsar, d_Tsma);
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, int p, int d, int q, int P, int D, int Q, int s,
                     int intercept, const double* d_params, double* loglike,
                     double* d_vs, bool trans, bool host_loglike, int fc_steps,
                     double* d_fc) {
  ML::PUSH_RANGE(__func__);

  // unpack parameters
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double *d_mu, *d_ar, *d_ma, *d_sar, *d_sma;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_ar, &d_ma,
                  &d_sar, &d_sma, intercept, &d_mu);
  unpack(d_params, d_mu, d_ar, d_ma, d_sar, d_sma, batch_size, p, q, P, Q,
         intercept, stream);

  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_mu, d_ar, d_ma, d_sar, d_sma, loglike, d_vs,
                  trans, host_loglike, fc_steps, d_fc);

  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_ar, d_ma,
                    d_sar, d_sma, intercept, d_mu);
  ML::POP_RANGE();
}

void information_criterion(cumlHandle& handle, const double* d_y,
                           int batch_size, int n_obs, int p, int d, int q,
                           int P, int D, int Q, int s, int intercept,
                           const double* d_mu, const double* d_ar,
                           const double* d_ma, const double* d_sar,
                           const double* d_sma, double* ic, int ic_type) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double* d_vs = (double*)allocator->allocate(
    sizeof(double) * (n_obs - d - s * D) * batch_size, stream);
  double* d_ic =
    (double*)allocator->allocate(sizeof(double) * batch_size, stream);

  /* Compute log-likelihood in d_ic */
  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_mu, d_ar, d_ma, d_sar, d_sma, d_ic, d_vs, false,
                  false);

  /* Compute information criterion from log-likelihood and base term */
  MLCommon::Metrics::Batched::information_criterion(
    d_ic, d_ic, static_cast<MLCommon::Metrics::IC_Type>(ic_type),
    p + q + P + Q + intercept, batch_size, n_obs - d - s * D, stream);

  /* Transfer information criterion device -> host */
  MLCommon::updateHost(ic, d_ic, batch_size, stream);

  allocator->deallocate(d_vs, sizeof(double) * (n_obs - d - s * D) * batch_size,
                        stream);
  allocator->deallocate(d_ic, sizeof(double) * batch_size, stream);
  ML::POP_RANGE();
}

/**
 * Auxiliary function of _start_params: least square approximation of an
 * ARMA model (with or without seasonality)
 * @note: in this function the non-seasonal case has s=1, not s=0!
 */
static void _arma_least_squares(
  cumlHandle& handle, double* d_ar, double* d_ma,
  const MLCommon::Matrix::BatchedMatrix<double>& bm_y, int p, int q,
  int s = 1) {
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  // Initialize params
  if (p)
    CUDA_CHECK(
      hipMemsetAsync(d_ar, 0, sizeof(double) * p * batch_size, stream));
  if (q)
    CUDA_CHECK(
      hipMemsetAsync(d_ma, 0, sizeof(double) * q * batch_size, stream));

  int ps = p * s, qs = q * s;
  int p_ar = 2 * qs;
  int r = std::max(p_ar + qs, ps);

  if ((q && p_ar >= n_obs - p_ar) || p + q >= n_obs - r) {
    // Too few observations for the estimate, keep 0
    return;
  }

  /* Matrix formed by lag matrices of y and the residuals respectively,
   * side by side. The left side will be used to estimate AR, the right
   * side to estimate MA */
  MLCommon::Matrix::BatchedMatrix<double> bm_ls_ar_res(
    n_obs - r, p + q, batch_size, cublas_handle, allocator, stream, false);
  int ar_offset = r - ps;
  int res_offset = (ps < p_ar + qs) ? 0 : ps - p_ar - qs;

  // Get residuals from an AR(p_ar) model to estimate the MA parameters
  if (q) {
    // Create lagged y
    int ls_height = n_obs - p_ar;
    MLCommon::Matrix::BatchedMatrix<double> bm_ls =
      MLCommon::Matrix::b_lagged_mat(bm_y, p_ar);

    /* Matrix for the initial AR fit, initialized by copy of y
     * (note: this is because gels works in-place ; the matrix has larger
     *  dimensions than the actual AR fit) */
    MLCommon::Matrix::BatchedMatrix<double> bm_ar_fit =
      MLCommon::Matrix::b_2dcopy(bm_y, p_ar, 0, ls_height, 1);

    // Residual, initialized as offset y to avoid one kernel call
    MLCommon::Matrix::BatchedMatrix<double> bm_residual(
      ls_height, 1, batch_size, cublas_handle, allocator, stream, false);
    MLCommon::copy(bm_residual.raw_data(), bm_ar_fit.raw_data(),
                   ls_height * batch_size, stream);

    // Initial AR fit
    MLCommon::Matrix::b_gels(bm_ls, bm_ar_fit);

    // Compute residual (technically a gemv)
    MLCommon::Matrix::b_gemm(false, false, ls_height, 1, p_ar, -1.0, bm_ls,
                             bm_ar_fit, 1.0, bm_residual);

    // Lags of the residual
    MLCommon::Matrix::b_lagged_mat(bm_residual, bm_ls_ar_res, q, n_obs - r,
                                   res_offset, (n_obs - r) * p, s);
  }

  // Lags of y
  MLCommon::Matrix::b_lagged_mat(bm_y, bm_ls_ar_res, p, n_obs - r, ar_offset, 0,
                                 s);

  /* Initializing the vector for the ARMA fit
   * (note: also in-place as described for AR fit) */
  MLCommon::Matrix::BatchedMatrix<double> bm_arma_fit =
    MLCommon::Matrix::b_2dcopy(bm_y, r, 0, n_obs - r, 1);

  // ARMA fit
  MLCommon::Matrix::b_gels(bm_ls_ar_res, bm_arma_fit);

  /* Copy the results in the AR and MA parameters batched vectors
   * Note: calling directly the kernel as there is not yet a way to wrap
   *       existing device pointers in a batched matrix */
  if (p) {
    MLCommon::Matrix::batched_2dcopy_kernel<<<batch_size, p, 0, stream>>>(
      bm_arma_fit.raw_data(), d_ar, 0, 0, n_obs - r, 1, p, 1);
    CUDA_CHECK(hipPeekAtLastError());
  }
  if (q) {
    MLCommon::Matrix::batched_2dcopy_kernel<<<batch_size, q, 0, stream>>>(
      bm_arma_fit.raw_data(), d_ma, p, 0, n_obs - r, 1, q, 1);
    CUDA_CHECK(hipPeekAtLastError());
  }
}

/**
 * Auxiliary function of estimate_x0: compute the starting parameters for
 * the series pre-processed by estimate_x0
 *
 * @note: bm_y can be mutated! estimate_x0 has already created a copy.
 */
static void _start_params(cumlHandle& handle, double* d_mu, double* d_ar,
                          double* d_ma, double* d_sar, double* d_sma,
                          MLCommon::Matrix::BatchedMatrix<double>& bm_y, int p,
                          int q, int P, int Q, int s, int intercept) {
  auto stream = handle.getStream();

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  if (intercept) {
    // Compute means and write them in mu
    MLCommon::Stats::mean(d_mu, bm_y.raw_data(), batch_size, n_obs, false,
                          false, stream);

    // Center the series around their means in-place
    MLCommon::LinAlg::matrixVectorOp(
      bm_y.raw_data(), bm_y.raw_data(), d_mu, batch_size, n_obs, false, true,
      [] __device__(double a, double b) { return a - b; }, stream);
  }

  // Estimate an ARMA fit without seasonality
  if (p + q) _arma_least_squares(handle, d_ar, d_ma, bm_y, p, q);

  // Estimate a seasonal ARMA fit independantly
  if (P + Q) _arma_least_squares(handle, d_sar, d_sma, bm_y, P, Q, s);
}

void estimate_x0(cumlHandle& handle, double* d_mu, double* d_ar, double* d_ma,
                 double* d_sar, double* d_sma, const double* d_y,
                 int batch_size, int n_obs, int p, int d, int q, int P, int D,
                 int Q, int s, int intercept) {
  ML::PUSH_RANGE(__func__);
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();

  // Difference if necessary, copy otherwise
  MLCommon::Matrix::BatchedMatrix<double> bm_yd(
    n_obs - d - s * D, 1, batch_size, cublas_handle, allocator, stream, false);
  _prepare_data(handle, bm_yd.raw_data(), d_y, batch_size, n_obs, d, D, s);
  // Note: mu is not known yet! We just want to difference the data

  // Do the computation of the initial parameters
  _start_params(handle, d_mu, d_ar, d_ma, d_sar, d_sma, bm_yd, p, q, P, Q, s,
                intercept);
  ML::POP_RANGE();
}

}  // namespace ML
