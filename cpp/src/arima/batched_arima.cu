#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include <common/nvtx.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include "arima_helpers.cuh"
#include "batched_arima.hpp"
#include "batched_kalman.hpp"

#include <common/cumlHandle.hpp>
#include <cuml/cuml.hpp>

#include <linalg/batched/batched_matrix.h>
#include <linalg/binary_op.h>
#include <linalg/cublas_wrappers.h>
#include <linalg/matrix_vector_op.h>
#include <linalg/unary_op.h>
#include <metrics/batched/information_criterion.h>
#include <stats/mean.h>
#include "cuda_utils.h"
#include "utils.h"

namespace ML {

/**
 * @brief Prepare data by differencing if needed (simple and/or seasonal)
 *        and removing a trend if needed
 *
 * @note: It is assumed that d + D <= 2. This is enforced on the Python side
 *
 * @param[in]  handle      cuML handle
 * @param[out] d_out       Output. Shape (n_obs - d - D*s, batch_size) (device)
 * @param[in]  d_in        Input. Shape (n_obs, batch_size) (device)
 * @param[in]  batch_size  Number of series per batch
 * @param[in]  n_obs       Number of observations per series
 * @param[in]  d           Order of simple differences (0, 1 or 1)
 * @param[in]  D           Order of seasonal differences (0, 1 or 1)
 * @param[in]  s           Seasonal period if D > 0
 * @param[in]  intercept   Whether the model fits an intercept
 * @param[in]  d_mu        Mu array if intercept > 0
 *                         Shape (batch_size,) (device)
 */
static void _prepare_data(cumlHandle& handle, double* d_out, const double* d_in,
                          int batch_size, int n_obs, int d, int D, int s,
                          int intercept = 0, const double* d_mu = nullptr) {
  const auto stream = handle.getStream();

  // Only one difference (simple or seasonal)
  if (d + D == 1) {
    int period = d ? 1 : s;
    int tpb = (n_obs - period) > 512 ? 256 : 128;  // quick heuristics
    MLCommon::LinAlg::Batched::
      batched_diff_kernel<<<batch_size, tpb, 0, stream>>>(d_in, d_out, n_obs,
                                                          period);
    CUDA_CHECK(hipPeekAtLastError());
  }
  // Two differences (simple or seasonal or both)
  else if (d + D == 2) {
    int period1 = d ? 1 : s;
    int period2 = d == 2 ? 1 : s;
    int tpb = (n_obs - period1 - period2) > 512 ? 256 : 128;
    MLCommon::LinAlg::Batched::
      batched_second_diff_kernel<<<batch_size, tpb, 0, stream>>>(
        d_in, d_out, n_obs, period1, period2);
    CUDA_CHECK(hipPeekAtLastError());
  }
  // If no difference and the pointers are different, copy in to out
  else if (d + D == 0 && d_in != d_out) {
    MLCommon::copy(d_out, d_in, n_obs * batch_size, stream);
  }
  // Other cases: no difference and the pointers are the same, nothing to do

  // Remove trend in-place
  if (intercept) {
    MLCommon::LinAlg::matrixVectorOp(
      d_out, d_out, d_mu, batch_size, n_obs - d - D * s, false, true,
      [] __device__(double a, double b) { return a - b; }, stream);
  }
}

/**
 * @brief Helper function that will read in src0 if the given index is
 *        negative, src1 otherwise.
 * @note  This is useful when one array is the logical continuation of
 *        another and the index is expressed relatively to the second array.
 */
static __device__ double _select_read(const double* src0, int size0,
                                      const double* src1, int idx) {
  return idx < 0 ? src0[size0 + idx] : src1[idx];
}

/**
 * @brief Kernel to undifference the data with up to two levels of simple
 *        and/or seasonal differencing.
 * @note  One thread per series.
 */
template <bool double_diff>
static __global__ void _undiff_kernel(double* d_fc, const double* d_in,
                                      int num_steps, int batch_size, int in_ld,
                                      int n_in, int s0, int s1 = 0) {
  int bid = blockIdx.x * blockDim.x + threadIdx.x;
  if (bid < batch_size) {
    double* b_fc = d_fc + bid * num_steps;
    const double* b_in = d_in + bid * in_ld;
    for (int i = 0; i < num_steps; i++) {
      if (!double_diff) {  // One simple or seasonal difference
        b_fc[i] += _select_read(b_in, n_in, b_fc, i - s0);
      } else {  // Two differences (simple, seasonal or both)
        double fc_acc = -_select_read(b_in, n_in, b_fc, i - s0 - s1);
        fc_acc += _select_read(b_in, n_in, b_fc, i - s0);
        fc_acc += _select_read(b_in, n_in, b_fc, i - s1);
        b_fc[i] += fc_acc;
      }
    }
  }
}

/**
 * @brief Finalizes a forecast by adding the trend and/or undifferencing
 *
 * @note: It is assumed that d + D <= 2. This is enforced on the Python side
 *
 * @param[in]     handle      cuML handle
 * @param[in|out] d_fc        Forecast. Shape (num_steps, batch_size) (device)
 * @param[in]     d_in        Original data. Shape (n_obs, batch_size) (device)
 * @param[in]     num_steps   Number of steps forecasted
 * @param[in]     batch_size  Number of series per batch
 * @param[in]     in_ld       Leading dimension of d_in
 * @param[in]     n_in        Number of observations/predictions in d_in
 * @param[in]     d           Order of simple differences (0, 1 or 1)
 * @param[in]     D           Order of seasonal differences (0, 1 or 1)
 * @param[in]     s           Seasonal period if D > 0
 * @param[in]     intercept   Whether the model fits an intercept
 * @param[in]     d_mu        Mu array if intercept > 0
 *                            Shape (batch_size,) (device)
 */
static void _finalize_forecast(cumlHandle& handle, double* d_fc,
                               const double* d_in, int num_steps,
                               int batch_size, int in_ld, int n_in, int d,
                               int D, int s, int intercept = 0,
                               const double* d_mu = nullptr) {
  const auto stream = handle.getStream();

  // Add the trend in-place
  if (intercept) {
    MLCommon::LinAlg::matrixVectorOp(
      d_fc, d_fc, d_mu, batch_size, num_steps, false, true,
      [] __device__(double a, double b) { return a + b; }, stream);
  }

  // Undifference
  constexpr int TPB = 64;  // One thread per series -> avoid big blocks
  if (d + D == 1) {
    _undiff_kernel<false>
      <<<MLCommon::ceildiv<int>(batch_size, TPB), TPB, 0, stream>>>(
        d_fc, d_in, num_steps, batch_size, in_ld, n_in, d ? 1 : s);
    CUDA_CHECK(hipPeekAtLastError());
  } else if (d + D == 2) {
    _undiff_kernel<true>
      <<<MLCommon::ceildiv<int>(batch_size, TPB), TPB, 0, stream>>>(
        d_fc, d_in, num_steps, batch_size, in_ld, n_in, d ? 1 : s,
        d == 2 ? 1 : s);
    CUDA_CHECK(hipPeekAtLastError());
  }
}

void residual(cumlHandle& handle, const double* d_y, int batch_size, int n_obs,
              int p, int d, int q, int P, int D, int Q, int s, int intercept,
              const double* d_mu, const double* d_ar, const double* d_ma,
              const double* d_sar, const double* d_sma, const double* d_sigma2,
              double* d_vs, bool trans, int fc_steps, double* d_fc) {
  ML::PUSH_RANGE(__func__);
  std::vector<double> loglike = std::vector<double>(batch_size);
  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_mu, d_ar, d_ma, d_sar, d_sma, d_sigma2,
                  loglike.data(), d_vs, trans, true, fc_steps, d_fc);
  ML::POP_RANGE();
}

void predict(cumlHandle& handle, const double* d_y, int batch_size, int n_obs,
             int start, int end, int p, int d, int q, int P, int D, int Q,
             int s, int intercept, const double* d_params, double* d_vs,
             double* d_y_p) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  const auto stream = handle.getStream();

  // Unpack parameters
  double *d_mu, *d_ar, *d_ma, *d_sar, *d_sma, *d_sigma2;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_ar, &d_ma,
                  &d_sar, &d_sma, &d_sigma2, false, intercept, &d_mu);
  unpack(d_params, d_mu, d_ar, d_ma, d_sar, d_sma, d_sigma2, batch_size, p, q,
         P, Q, intercept, stream);

  // Prepare data
  int d_sD = d + D * s;
  int ld_yprep = n_obs - d_sD;
  double* d_y_prep = (double*)allocator->allocate(
    ld_yprep * batch_size * sizeof(double), stream);
  _prepare_data(handle, d_y_prep, d_y, batch_size, n_obs, d, D, s, intercept,
                d_mu);

  // Create temporary array for the forecasts
  int num_steps = std::max(end - n_obs, 0);
  double* d_y_fc = nullptr;
  if (num_steps) {
    d_y_fc = (double*)allocator->allocate(
      num_steps * batch_size * sizeof(double), stream);
  }

  // Compute the residual and forecast - provide already prepared data and
  // extracted parameters
  residual(handle, d_y_prep, batch_size, n_obs - d_sD, p, 0, q, P, 0, Q, s, 0,
           nullptr, d_ar, d_ma, d_sar, d_sma, d_sigma2, d_vs, false, num_steps,
           d_y_fc);

  auto counting = thrust::make_counting_iterator(0);
  int predict_ld = end - start;

  //
  // In-sample prediction
  //

  int p_start = std::max(start, d_sD);
  int p_end = std::min(n_obs, end);

  // The prediction loop starts by filling undefined predictions with NaN,
  // then computes the predictions from the observations and residuals
  if (start < n_obs) {
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_y_p[0] = 0.0;
                       for (int i = 0; i < d_sD - start; i++) {
                         d_y_p[bid * predict_ld + i] = nan("");
                       }
                       for (int i = p_start; i < p_end; i++) {
                         d_y_p[bid * predict_ld + i - start] =
                           d_y[bid * n_obs + i] -
                           d_vs[bid * ld_yprep + i - d_sD];
                       }
                     });
  }

  //
  // Finalize out-of-sample forecast and copy in-sample predictions
  //

  if (num_steps) {
    // Add trend and/or undiff
    _finalize_forecast(handle, d_y_fc, d_y, num_steps, batch_size, n_obs, n_obs,
                       d, D, s, intercept, d_mu);

    // Copy forecast in d_y_p
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       for (int i = 0; i < num_steps; i++) {
                         d_y_p[bid * predict_ld + n_obs - start + i] =
                           d_y_fc[num_steps * bid + i];
                       }
                     });

    allocator->deallocate(d_y_fc, num_steps * batch_size * sizeof(double),
                          stream);
  }

  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_ar, d_ma,
                    d_sar, d_sma, d_sigma2, false, intercept, d_mu);
  allocator->deallocate(d_y_prep, ld_yprep * batch_size * sizeof(double),
                        stream);
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, int p, int d, int q, int P, int D, int Q, int s,
                     int intercept, const double* d_mu, const double* d_ar,
                     const double* d_ma, const double* d_sar,
                     const double* d_sma, const double* d_sigma2,
                     double* loglike, double* d_vs, bool trans,
                     bool host_loglike, int fc_steps, double* d_fc) {
  ML::PUSH_RANGE(__func__);

  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double *d_Tar, *d_Tma, *d_Tsar, *d_Tsma;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_Tar, &d_Tma,
                  &d_Tsar, &d_Tsma, nullptr, true);

  if (trans) {
    batched_jones_transform(handle, p, q, P, Q, batch_size, false, d_ar, d_ma,
                            d_sar, d_sma, d_Tar, d_Tma, d_Tsar, d_Tsma);
  } else {
    // non-transformed case: just use original parameters
    CUDA_CHECK(hipMemcpyAsync(d_Tar, d_ar, sizeof(double) * batch_size * p,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tma, d_ma, sizeof(double) * batch_size * q,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tsar, d_sar, sizeof(double) * batch_size * P,
                               hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Tsma, d_sma, sizeof(double) * batch_size * Q,
                               hipMemcpyDeviceToDevice, stream));
  }

  if (d + D + intercept == 0) {
    batched_kalman_filter(handle, d_y, n_obs, d_Tar, d_Tma, d_Tsar, d_Tsma,
                          d_sigma2, p, q, P, Q, s, batch_size, loglike, d_vs,
                          host_loglike, false, fc_steps, d_fc);
  } else {
    double* d_y_prep = (double*)allocator->allocate(
      batch_size * (n_obs - d - s * D) * sizeof(double), stream);

    _prepare_data(handle, d_y_prep, d_y, batch_size, n_obs, d, D, s, intercept,
                  d_mu);

    batched_kalman_filter(handle, d_y_prep, n_obs - d - s * D, d_Tar, d_Tma,
                          d_Tsar, d_Tsma, d_sigma2, p, q, P, Q, s, batch_size,
                          loglike, d_vs, host_loglike, false, fc_steps, d_fc);

    allocator->deallocate(
      d_y_prep, sizeof(double) * batch_size * (n_obs - d - s * D), stream);
  }
  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_Tar, d_Tma,
                    d_Tsar, d_Tsma, nullptr, true);
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, int p, int d, int q, int P, int D, int Q, int s,
                     int intercept, const double* d_params, double* loglike,
                     double* d_vs, bool trans, bool host_loglike, int fc_steps,
                     double* d_fc) {
  ML::PUSH_RANGE(__func__);

  // unpack parameters
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double *d_mu, *d_ar, *d_ma, *d_sar, *d_sma, *d_sigma2;
  allocate_params(allocator, stream, p, q, P, Q, batch_size, &d_ar, &d_ma,
                  &d_sar, &d_sma, &d_sigma2, false, intercept, &d_mu);
  unpack(d_params, d_mu, d_ar, d_ma, d_sar, d_sma, d_sigma2, batch_size, p, q,
         P, Q, intercept, stream);

  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_mu, d_ar, d_ma, d_sar, d_sma, d_sigma2, loglike,
                  d_vs, trans, host_loglike, fc_steps, d_fc);

  deallocate_params(allocator, stream, p, q, P, Q, batch_size, d_ar, d_ma,
                    d_sar, d_sma, d_sigma2, false, intercept, d_mu);
  ML::POP_RANGE();
}

void information_criterion(cumlHandle& handle, const double* d_y,
                           int batch_size, int n_obs, int p, int d, int q,
                           int P, int D, int Q, int s, int intercept,
                           const double* d_mu, const double* d_ar,
                           const double* d_ma, const double* d_sar,
                           const double* d_sma, const double* d_sigma2,
                           double* ic, int ic_type) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double* d_vs = (double*)allocator->allocate(
    sizeof(double) * (n_obs - d - s * D) * batch_size, stream);
  double* d_ic =
    (double*)allocator->allocate(sizeof(double) * batch_size, stream);

  /* Compute log-likelihood in d_ic */
  batched_loglike(handle, d_y, batch_size, n_obs, p, d, q, P, D, Q, s,
                  intercept, d_mu, d_ar, d_ma, d_sar, d_sma, d_sigma2, d_ic,
                  d_vs, false, false);

  /* Compute information criterion from log-likelihood and base term */
  MLCommon::Metrics::Batched::information_criterion(
    d_ic, d_ic, static_cast<MLCommon::Metrics::IC_Type>(ic_type),
    p + q + P + Q + intercept + 1, batch_size, n_obs - d - s * D, stream);

  /* Transfer information criterion device -> host */
  MLCommon::updateHost(ic, d_ic, batch_size, stream);

  allocator->deallocate(d_vs, sizeof(double) * (n_obs - d - s * D) * batch_size,
                        stream);
  allocator->deallocate(d_ic, sizeof(double) * batch_size, stream);
  ML::POP_RANGE();
}

/**
 * Auxiliary function of _start_params: least square approximation of an
 * ARMA model (with or without seasonality)
 * @note: in this function the non-seasonal case has s=1, not s=0!
 */
static void _arma_least_squares(
  cumlHandle& handle, double* d_ar, double* d_ma, double* d_sigma2,
  const MLCommon::LinAlg::Batched::BatchedMatrix<double>& bm_y, int p, int q,
  int s, bool estimate_sigma2, int k = 0, double* d_mu = nullptr) {
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  int ps = p * s, qs = q * s;
  int p_ar = 2 * qs;
  int r = std::max(p_ar + qs, ps);

  if ((q && p_ar >= n_obs - p_ar) || p + q + k >= n_obs - r) {
    // Too few observations for the estimate, fill with 0
    if (k)
      CUDA_CHECK(hipMemsetAsync(d_mu, 0, sizeof(double) * batch_size, stream));
    if (p)
      CUDA_CHECK(
        hipMemsetAsync(d_ar, 0, sizeof(double) * p * batch_size, stream));
    if (q)
      CUDA_CHECK(
        hipMemsetAsync(d_ma, 0, sizeof(double) * q * batch_size, stream));
    return;
  }

  /* Matrix formed by lag matrices of y and the residuals respectively,
   * side by side. The left side will be used to estimate AR, the right
   * side to estimate MA */
  MLCommon::LinAlg::Batched::BatchedMatrix<double> bm_ls_ar_res(
    n_obs - r, p + q + k, batch_size, cublas_handle, allocator, stream, false);
  int ar_offset = r - ps;
  int res_offset = r - p_ar - qs;

  // Get residuals from an AR(p_ar) model to estimate the MA parameters
  if (q) {
    // Create lagged y
    int ls_height = n_obs - p_ar;
    MLCommon::LinAlg::Batched::BatchedMatrix<double> bm_ls =
      MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, p_ar);

    /* Matrix for the initial AR fit, initialized by copy of y
     * (note: this is because gels works in-place ; the matrix has larger
     *  dimensions than the actual AR fit) */
    MLCommon::LinAlg::Batched::BatchedMatrix<double> bm_ar_fit =
      MLCommon::LinAlg::Batched::b_2dcopy(bm_y, p_ar, 0, ls_height, 1);

    // Residual, initialized as offset y to avoid one kernel call
    MLCommon::LinAlg::Batched::BatchedMatrix<double> bm_residual(bm_ar_fit);

    // Initial AR fit
    MLCommon::LinAlg::Batched::b_gels(bm_ls, bm_ar_fit);

    // Compute residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false, false, ls_height, 1, p_ar, -1.0,
                                      bm_ls, bm_ar_fit, 1.0, bm_residual);

    // Lags of the residual
    MLCommon::LinAlg::Batched::b_lagged_mat(bm_residual, bm_ls_ar_res, q,
                                            n_obs - r, res_offset,
                                            (n_obs - r) * (k + p), s);
  }

  // Fill the first column of the matrix with 1 if we fit an intercept
  auto counting = thrust::make_counting_iterator(0);
  if (k) {
    double* d_ls_ar_res = bm_ls_ar_res.raw_data();
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       double* b_ls_ar_res =
                         d_ls_ar_res + bid * (n_obs - r) * (p + q + k);
                       for (int i = 0; i < n_obs - r; i++) {
                         b_ls_ar_res[i] = 1.0;
                       }
                     });
  }

  // Lags of y
  MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, bm_ls_ar_res, p, n_obs - r,
                                          ar_offset, (n_obs - r) * k, s);

  /* Initializing the vector for the ARMA fit
   * (note: also in-place as described for AR fit) */
  MLCommon::LinAlg::Batched::BatchedMatrix<double> bm_arma_fit =
    MLCommon::LinAlg::Batched::b_2dcopy(bm_y, r, 0, n_obs - r, 1);

  // The residuals will be computed only if sigma2 is requested
  MLCommon::LinAlg::Batched::BatchedMatrix<double> bm_final_residual(
    n_obs - r, 1, batch_size, cublas_handle, allocator, stream, false);
  if (estimate_sigma2) {
    MLCommon::copy(bm_final_residual.raw_data(), bm_arma_fit.raw_data(),
                   (n_obs - r) * batch_size, stream);
  }

  // ARMA fit
  MLCommon::LinAlg::Batched::b_gels(bm_ls_ar_res, bm_arma_fit);

  // Copy the results in the parameter vectors
  const double* d_arma_fit = bm_arma_fit.raw_data();
  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     const double* b_arma_fit = d_arma_fit + bid * (n_obs - r);
                     if (k) {
                       d_mu[bid] = b_arma_fit[0];
                     }
                     if (p) {
                       double* b_ar = d_ar + bid * p;
                       for (int i = 0; i < p; i++) {
                         b_ar[i] = b_arma_fit[i + k];
                       }
                     }
                     if (q) {
                       double* b_ma = d_ma + bid * q;
                       for (int i = 0; i < q; i++) {
                         b_ma[i] = b_arma_fit[i + p + k];
                       }
                     }
                   });

  if (estimate_sigma2) {
    // Compute final residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false, false, n_obs - r, 1, p + q + k,
                                      -1.0, bm_ls_ar_res, bm_arma_fit, 1.0,
                                      bm_final_residual);

    // Compute variance
    double* d_residual = bm_final_residual.raw_data();
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       double acc = 0.0;
                       const double* b_residual =
                         d_residual + (n_obs - r) * bid;
                       for (int i = q; i < n_obs - r; i++) {
                         double res = b_residual[i];
                         acc += res * res;
                       }
                       d_sigma2[bid] = acc / static_cast<double>(n_obs - r - q);
                     });
  }
}

/**
 * Auxiliary function of estimate_x0: compute the starting parameters for
 * the series pre-processed by estimate_x0
 */
static void _start_params(
  cumlHandle& handle, double* d_mu, double* d_ar, double* d_ma, double* d_sar,
  double* d_sma, double* d_sigma2,
  const MLCommon::LinAlg::Batched::BatchedMatrix<double>& bm_y, int p, int q,
  int P, int Q, int s, int k) {
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto allocator = handle_impl.getDeviceAllocator();

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  // Estimate an ARMA fit without seasonality
  if (p + q + k)
    _arma_least_squares(handle, d_ar, d_ma, d_sigma2, bm_y, p, q, 1, true, k,
                        d_mu);

  // Estimate a seasonal ARMA fit independantly
  if (P + Q)
    _arma_least_squares(handle, d_sar, d_sma, d_sigma2, bm_y, P, Q, s,
                        p + q + k == 0);
}

void estimate_x0(cumlHandle& handle, double* d_mu, double* d_ar, double* d_ma,
                 double* d_sar, double* d_sma, double* d_sigma2,
                 const double* d_y, int batch_size, int n_obs, int p, int d,
                 int q, int P, int D, int Q, int s, int intercept) {
  ML::PUSH_RANGE(__func__);
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();

  // Difference if necessary, copy otherwise
  MLCommon::LinAlg::Batched::BatchedMatrix<double> bm_yd(
    n_obs - d - s * D, 1, batch_size, cublas_handle, allocator, stream, false);
  _prepare_data(handle, bm_yd.raw_data(), d_y, batch_size, n_obs, d, D, s);
  // Note: mu is not known yet! We just want to difference the data

  // Do the computation of the initial parameters
  _start_params(handle, d_mu, d_ar, d_ma, d_sar, d_sma, d_sigma2, bm_yd, p, q,
                P, Q, s, intercept);
  ML::POP_RANGE();
}

}  // namespace ML
