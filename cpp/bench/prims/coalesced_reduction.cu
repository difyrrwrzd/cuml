/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "benchmark.cuh"
#include <linalg/coalesced_reduction.h>

namespace MLCommon {
namespace Bench {
namespace LinAlg {

struct Params {
  int rows, cols;
};  // struct Params

template <typename T>
struct CoalescedReduction : public Fixture {
  CoalescedReduction(const std::string& name, const Params& p) : Fixture(name),
                                                                 params(p) {}

 protected:
  void allocateBuffers(const ::benchmark::State& state) override {
    allocate(data, params.rows * params.cols, true);
    allocate(dots, params.rows, true);
  }

  void deallocateBuffers(const ::benchmark::State& state) override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(dots));
  }

  void runBenchmark(::benchmark::State& state) override {
    for (auto _ : state) {
      CudaEventTimer timer(state, true, stream);
      MLCommon::LinAlg::coalescedReduction(dots, data, params.cols, params.rows,
                                           T(0.f), stream);
    }
  }

 private:
  Params params;
  T *data, *dots;
};  // struct CoalescedReduction

static std::vector<Params> getInputs() {
  return {
    {8 * 1024, 1024},
    {1024, 8 * 1024},
    {8 * 1024, 8 * 1024},
    {32 * 1024, 1024},
    {1024, 32 * 1024},
    {32 * 1024, 32 * 1024},
  };
}

PRIMS_BENCH_REGISTER(Params, CoalescedReduction<float>, "coalescedReduction",
                     getInputs());
PRIMS_BENCH_REGISTER(Params, CoalescedReduction<double>, "coalescedReduction",
                     getInputs());

}  // namespace LinAlg
}  // namespace Bench
}  // namespace MLCommon
