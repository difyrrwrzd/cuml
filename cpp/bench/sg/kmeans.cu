/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuML.hpp>
#include <kmeans/kmeans.hpp>
#include "dataset.h"
#include "harness.h"

namespace ML {
namespace Bench {
namespace kmeans {

template <typename D>
struct Params : public DatasetParams {
  // dataset generation related
  D cluster_std;
  bool shuffle;
  D center_box_min, center_box_max;
  uint64_t seed;
  // algo related
  ML::kmeans::KMeansParams p;

  std::string str() const {
    std::ostringstream oss;
    oss << ";cluster_std=" << cluster_std << ";shuffle=" << shuffle
        << ";center_box_min=" << center_box_min
        << ";center_box_max=" << center_box_max << ";seed=" << seed
        << ";init=" << p.init << ";max_iter=" << p.max_iter << ";tol=" << p.tol
        << ";verbose=" << p.verbose << ";metric=" << p.metric
        << ";oversampling_factor=" << p.oversampling_factor
        << ";batch_size=" << p.batch_size
        << ";inertia-check=" << p.inertia_check;
    return DatasetParams::str() + oss.str();
  }
};

template <typename D>
struct Run : public Benchmark<Params<D>> {
  void setup() {
    const auto& p = this->getParams();
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.reset(new cumlHandle);
    handle->setStream(stream);
    auto allocator = handle->getDeviceAllocator();
    labels = (int*)allocator->allocate(p.nrows * sizeof(int), stream);
    centroids =
      (D*)allocator->allocate(p.nclasses * p.ncols * sizeof(D), stream);
    dataset.blobs(*handle, p.nrows, p.ncols, p.rowMajor, p.nclasses,
                  p.cluster_std, p.shuffle, p.center_box_min, p.center_box_max,
                  p.seed);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void teardown() {
    const auto& p = this->getParams();
    CUDA_CHECK(hipStreamSynchronize(stream));
    auto allocator = handle->getDeviceAllocator();
    allocator->deallocate(labels, p.nrows * sizeof(int), stream);
    allocator->deallocate(centroids, p.nclasses * p.ncols * sizeof(D), stream);
    dataset.deallocate(*handle);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  ///@todo: implement
  void metrics(RunInfo& ri) {}

  void run() {
    const auto& p = this->getParams();
    ASSERT(p.rowMajor, "Kmeans only supports row-major inputs");
    ML::kmeans::fit_predict(*handle, p.p, dataset.X, p.nrows, p.ncols,
                            centroids, labels, inertia, nIter);
    CUDA_CHECK(hipStreamSynchronize(handle->getStream()));
  }

 private:
  std::shared_ptr<cumlHandle> handle;
  hipStream_t stream;
  int* labels;
  D* centroids;
  Dataset<D, int> dataset;
  int nIter;
  D inertia;
};

template <typename D>
std::vector<Params<D>> getInputs() {
  std::vector<Params<D>> out;
  Params<D> p;
  p.rowMajor = true;
  p.cluster_std = (D)1.0;
  p.shuffle = false;
  p.center_box_min = (D)-10.0;
  p.center_box_max = (D)10.0;
  p.seed = 12345ULL;
  p.p.init = (ML::kmeans::KMeansParams::InitMethod)0;
  p.p.max_iter = 300;
  p.p.tol = (D)1e-4;
  p.p.verbose = false;
  p.p.seed = p.seed;
  p.p.metric = 0;  // L2
  p.p.inertia_check = true;
  std::vector<std::pair<int, int>> rowcols = {
    {40000, 128},
    {80000, 128},
    {160000, 128},
  };
  for (auto& rc : rowcols) {
    p.nrows = rc.first;
    p.ncols = rc.second;
    for (auto nclass : std::vector<int>({8, 16, 32})) {
      p.nclasses = nclass;
      p.p.n_clusters = p.nclasses;
      for (auto bs_shift : std::vector<int>({16, 18, 20})) {
        p.p.batch_size = 1 << bs_shift;
        out.push_back(p);
      }
    }
  }
  return out;
}

REGISTER_BENCH(Run<float>, Params<float>, kmeansF, getInputs<float>());
REGISTER_BENCH(Run<double>, Params<double>, kmeansD, getInputs<double>());

}  // end namespace kmeans
}  // end namespace Bench
}  // end namespace ML
