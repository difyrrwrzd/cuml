/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuML.hpp>
#include <kmeans/kmeans.hpp>
#include "dataset.h"
#include "harness.h"

namespace ML {
namespace Bench {
namespace kmeans {

template <typename D>
struct Params : public DatasetParams {
  // dataset generation related
  D cluster_std;
  bool shuffle;
  D center_box_min, center_box_max;
  uint64_t seed;
  // algo related
  kmeans::KMeansParams p;

  Params() : DatasetParams() {}

  Params(int nr, int nc, int ncl, bool row, D std, bool shfl, D cmin, D cmax,
         uint64_t s, int ini, int mi, double tol, bool ver, int mt, int of,
         int bs, bool ic)
    : DatasetParams(nr, nc, ncl, row),
      cluster_std(std),
      shuffle(shfl),
      center_box_min(cmin),
      center_box_max(cmax),
      seed(s),
      p() {
    p.n_clusters = nclasses;
    p.init = (kmeans::KMeansParams::InitMethod)ini;
    p.max_iter = mi;
    p.tol = tol;
    p.verbose = ver;
    p.seed = seed;
    p.metric = mt;
    p.oversampling_factor = of;
    p.batch_size = bs;
    p.interia_check = ic;
  }

  std::string str() const {
    std::ostringstream oss;
    oss << ";cluster_std=" << cluster_std << ";shuffle=" << shuffle
        << ";center_box_min=" << center_box_min
        << ";center_box_max=" << center_box_max << ";seed=" << seed
        << ";init=" << p.init << ";max_iter=" << p.max_iter
        << ";tol=" << p.tol << ";verbose=" << verbose << ";metric=" << p.metric
        << ";oversampling_factor=" << p.oversampling_factor << ";batch_size="
        << p.batch_size << ";inertia-check=" << p.inertia_check;
    return DatasetParams::str() + oss.str();
  }
};

template <typename D>
struct Run : public Benchmark<Params<D>> {
  void setup() {
    const auto& p = this->getParams();
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.reset(new cumlHandle);
    handle->setStream(stream);
    auto allocator = handle->getDeviceAllocator();
    labels = (int*)allocator->allocate(p.nrows * sizeof(int), stream);
    centroids = (D*)allocator->allocate(p.nclasses * p.ncols * sizeof(D), stream);
    dataset.blobs(*handle, p.nrows, p.ncols, p.rowMajor, p.nclasses,
                  p.cluster_std, p.shuffle, p.center_box_min, p.center_box_max,
                  p.seed);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void teardown() {
    const auto& p = this->getParams();
    CUDA_CHECK(hipStreamSynchronize(stream));
    auto allocator = handle->getDeviceAllocator();
    allocator->deallocate(labels, p.nrows * sizeof(int), stream);
    allocator->deallocate(centroids, p.nclasses * p.ncols * sizeof(D), stream);
    dataset.deallocate(*handle);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  ///@todo: implement
  void metrics(RunInfo& ri) {}

  void run() {
    const auto& p = this->getParams();
    ASSERT(p.rowMajor, "Kmeans only supports row-major inputs");
    kmeans::fit_predict(*handle, p.p, dataset.X, p.nrows, p.ncols, centroids,
                        labels, inertia, nIter);
    CUDA_CHECK(hipStreamSynchronize(handle->getStream()));
  }

 private:
  std::shared_ptr<cumlHandle> handle;
  hipStream_t stream;
  int* labels;
  D* centroids;
  Dataset<D, int> dataset;
  int nIter;
  D inertia;
};

template <typename D>
std::vector<Params<D>> getInputs() {
  std::vector<Params<D>> out;
  Params<D> p;
  p.rowMajor = true;
  p.cluster_std = (D)1.0;
  p.shuffle = false;
  p.center_box_min = (D)-10.0;
  p.center_box_max = (D)10.0;
  p.seed = 12345ULL;
  p.max_bytes_per_batch = 0;
  std::vector<std::pair<int, int>> rowcols = {
    {10000, 81},
    {20000, 128},
    {40000, 128},
  };
  for (auto& rc : rowcols) {
    p.nrows = rc.first;
    p.ncols = rc.second;
    for (auto nclass : std::vector<int>({2, 4, 8})) {
      p.nclasses = nclass;
      for (auto ep : std::vector<D>({0.1, 1.0})) {
        p.eps = ep;
        for (auto mp : std::vector<int>({3, 10})) {
          p.min_pts = mp;
          out.push_back(p);
        }
      }
    }
  }
  return out;
}

REGISTER_BENCH(Run<float>, Params<float>, kmeansF, getInputs<float>());
REGISTER_BENCH(Run<double>, Params<double>, kmeansD, getInputs<double>());

}  // end namespace kmeans
}  // end namespace Bench
}  // end namespace ML
